#include "hip/hip_runtime.h"
/*
* Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
*
* NVIDIA Corporation and its licensors retain all intellectual property and
* proprietary rights in and to this software and related documentation.
* Any use, reproduction, disclosure, or distribution of this software
* and related documentation without an express license agreement from
* NVIDIA Corporation is strictly prohibited.
*
* Please refer to the applicable NVIDIA end user license agreement (EULA)
* associated with this source code for terms and conditions that govern
* your use of this NVIDIA software.
*
*/
#include <iostream>
#include "string"
#include "fstream"
#include "cudaFEM_read.cuh"

#include <stdio.h> 
#include <stdlib.h>
#include <hip/hip_runtime.h> 
#include "hipblas.h" 
#include <iostream>
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>


#include <hipfft/hipfft.h>
#include "fstream"

#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include ""
#include <hipsolver.h>
#include <hipsparse.h>
#include <vector>
#include <cassert>
#include "Utilities.cuh"
#include <ctime>

#include "cuda_functions.cuh"
#define nodesinelemX(node,el,nodesPerElem) (node + nodesPerElem*el)
#define threeD21D(row_d,col_d,el_d,width_d,depth_d) (row_d+width_d*(col_d+depth_d*el_d))
#define nodesDisplacementX(dof,node,dimension) (dof + node*dimension)
#define IDX2C(i,j,ld) (((j)*(ld))+( i )) 

//This is for the local K matrix
//NOTE:::: nu and E are not initilized
__device__ inline float atomicAdda(float* address, double value)

{

	float ret = atomicExch(address, 0.0f);

	float old = ret + (float) value;

	while ((old = atomicExch(address, old)) != 0.0f)

	{

		old = atomicExch(address, 0.0f) + old;

	}

	return ret;

};
__global__ void make_K_cuda(double *E_vector, int *nodesInElem, double *x_vector, double *y_vector, double *z_vector, int *displaceInElem_device, float *d_A_dense,int numnodes) {
	//int x = threadIdx.x + blockIdx.x*blockDim.x; //if we have a 3D problem then this will go from 0 to 11
	int row;
	int dummy_node;
	int loop_node;
	int dummy_row;
	int dummy_col;
	int DOF[12];
	int counter;
	int offset = threadIdx.x + blockIdx.x*blockDim.x; // offset will essentaillay be the element counter
	int max_limit = 12 * 12 * 4374;
	double E = 200000;
	double nu = 0.45;
	double x14 = x_vector[nodesInElem[nodesinelemX(0, offset, 4)]] - x_vector[nodesInElem[nodesinelemX(3, offset, 4)]];
	double x24 = x_vector[nodesInElem[nodesinelemX(1, offset, 4)]] - x_vector[nodesInElem[nodesinelemX(3, offset, 4)]];
	double x34 = x_vector[nodesInElem[nodesinelemX(2, offset, 4)]] - x_vector[nodesInElem[nodesinelemX(3, offset, 4)]];
	double y14 = y_vector[nodesInElem[nodesinelemX(0, offset, 4)]] - y_vector[nodesInElem[nodesinelemX(3, offset, 4)]];
	double y24 = y_vector[nodesInElem[nodesinelemX(1, offset, 4)]] - y_vector[nodesInElem[nodesinelemX(3, offset, 4)]];
	double y34 = y_vector[nodesInElem[nodesinelemX(2, offset, 4)]] - y_vector[nodesInElem[nodesinelemX(3, offset, 4)]];
	double z14 = z_vector[nodesInElem[nodesinelemX(0, offset, 4)]] - z_vector[nodesInElem[nodesinelemX(3, offset, 4)]];
	double z24 = z_vector[nodesInElem[nodesinelemX(1, offset, 4)]] - z_vector[nodesInElem[nodesinelemX(3, offset, 4)]];
	double z34 = z_vector[nodesInElem[nodesinelemX(2, offset, 4)]] - z_vector[nodesInElem[nodesinelemX(3, offset, 4)]];

	//std::cout << x14*(y24*z34 - y34*z24) - y14*(x24*z34 - z24 * 34) + z14*(x24*y34 - y24*x34) << std::endl;
	double det_J = (x14*(y24*z34 - y34*z24) - y14*(x24*z34 - z24 * x34) + z14*(x24*y34 - y24*x34));

	double J_bar11 = (y24*z34 - z24*y34) / det_J;
	double J_bar12 = (z14*y34 - y14*z34) / det_J;
	double J_bar13 = (y14*z24 - z14*y24) / det_J;
	double J_bar21 = (z24*x34 - x24*z34) / det_J;
	double J_bar22 = (x14*z34 - z14*x34) / det_J;
	double J_bar23 = (z14*x24 - x14*z24) / det_J;
	double J_bar31 = (x24*y34 - y24*x34) / det_J;
	double J_bar32 = (y14*x34 - x14*y34) / det_J;
	double J_bar33 = (x14*y24 - y14*x24) / det_J;

	double J_star1 = -(J_bar11 + J_bar12 + J_bar13);
	double J_star2 = -(J_bar21 + J_bar22 + J_bar23);
	double J_star3 = -(J_bar31 + J_bar32 + J_bar33);

	
	
		E_vector[offset*144 + 0] = 0.166666666666667*E*J_bar11*J_bar11 * det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar21 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar31 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 1] = 0.166666666666667*E*J_bar11*J_bar21*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar11*J_bar21*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 2] = 0.166666666666667*E*J_bar11*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar11*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 3] = 0.166666666666667*E*J_bar11*J_bar12*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar22*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 4] = 0.166666666666667*E*J_bar11*J_bar22*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar21*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 5] = 0.166666666666667*E*J_bar11*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 6] = 0.166666666666667*E*J_bar11*J_bar13*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 7] = 0.166666666666667*E*J_bar11*J_bar23*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar21*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 8] = 0.166666666666667*E*J_bar11*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 9] = 0.166666666666667*E*J_bar11*J_star1*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 10] = 0.166666666666667*E*J_bar11*J_star2*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 11] = 0.166666666666667*E*J_bar11*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 12] = 0.166666666666667*E*J_bar11*J_bar21*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar11*J_bar21*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 13] = 0.166666666666667*E*J_bar11*J_bar11 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar21 * det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar31 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 14] = 0.166666666666667*E*J_bar21*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 15] = 0.166666666666667*E*J_bar11*J_bar22*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar21*det_J*nu / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 16] = 0.166666666666667*E*J_bar11*J_bar12*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar22*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 17] = 0.166666666666667*E*J_bar21*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 18] = 0.166666666666667*E*J_bar11*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar21*det_J*nu / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 19] = 0.166666666666667*E*J_bar11*J_bar13*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar23*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 20] = 0.166666666666667*E*J_bar21*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 21] = 0.166666666666667*E*J_bar11*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_star1*det_J*nu / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 22] = 0.166666666666667*E*J_bar11*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_star2*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 23] = 0.166666666666667*E*J_bar21*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 24] = 0.166666666666667*E*J_bar11*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar11*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 25] = 0.166666666666667*E*J_bar21*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 26] = 0.166666666666667*E*J_bar11*J_bar11 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar21 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar31 * det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 27] = 0.166666666666667*E*J_bar11*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 28] = 0.166666666666667*E*J_bar21*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 29] = 0.166666666666667*E*J_bar11*J_bar12*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar22*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar32*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 30] = 0.166666666666667*E*J_bar11*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 31] = 0.166666666666667*E*J_bar21*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 32] = 0.166666666666667*E*J_bar11*J_bar13*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar33*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 33] = 0.166666666666667*E*J_bar11*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star1*det_J*nu / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 34] = 0.166666666666667*E*J_bar21*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star2*det_J*nu / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 35] = 0.166666666666667*E*J_bar11*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star3*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 36] = 0.166666666666667*E*J_bar11*J_bar12*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar22*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 37] = 0.166666666666667*E*J_bar11*J_bar22*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar21*det_J*nu / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 38] = 0.166666666666667*E*J_bar11*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 39] = 0.166666666666667*E*J_bar12*J_bar12 * det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar22 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_bar32 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 40] = 0.166666666666667*E*J_bar12*J_bar22*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar22*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 41] = 0.166666666666667*E*J_bar12*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 42] = 0.166666666666667*E*J_bar12*J_bar13*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 43] = 0.166666666666667*E*J_bar12*J_bar23*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar22*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 44] = 0.166666666666667*E*J_bar12*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 45] = 0.166666666666667*E*J_bar12*J_star1*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 46] = 0.166666666666667*E*J_bar12*J_star2*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 47] = 0.166666666666667*E*J_bar12*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 48] = 0.166666666666667*E*J_bar11*J_bar22*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar21*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 49] = 0.166666666666667*E*J_bar11*J_bar12*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar22*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 50] = 0.166666666666667*E*J_bar21*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 51] = 0.166666666666667*E*J_bar12*J_bar22*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar22*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 52] = 0.166666666666667*E*J_bar12*J_bar12 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar22 * det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_bar32 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 53] = 0.166666666666667*E*J_bar22*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 54] = 0.166666666666667*E*J_bar12*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar22*det_J*nu / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 55] = 0.166666666666667*E*J_bar12*J_bar13*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar23*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 56] = 0.166666666666667*E*J_bar22*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 57] = 0.166666666666667*E*J_bar12*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_star1*det_J*nu / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 58] = 0.166666666666667*E*J_bar12*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_star2*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 59] = 0.166666666666667*E*J_bar22*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 60] = 0.166666666666667*E*J_bar11*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 61] = 0.166666666666667*E*J_bar21*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 62] = 0.166666666666667*E*J_bar11*J_bar12*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar22*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar32*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 63] = 0.166666666666667*E*J_bar12*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 64] = 0.166666666666667*E*J_bar22*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 65] = 0.166666666666667*E*J_bar12*J_bar12 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar22 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_bar32 * det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 66] = 0.166666666666667*E*J_bar12*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 67] = 0.166666666666667*E*J_bar22*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 68] = 0.166666666666667*E*J_bar12*J_bar13*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_bar33*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 69] = 0.166666666666667*E*J_bar12*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star1*det_J*nu / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 70] = 0.166666666666667*E*J_bar22*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star2*det_J*nu / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 71] = 0.166666666666667*E*J_bar12*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star3*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 72] = 0.166666666666667*E*J_bar11*J_bar13*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 73] = 0.166666666666667*E*J_bar11*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar21*det_J*nu / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 74] = 0.166666666666667*E*J_bar11*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 75] = 0.166666666666667*E*J_bar12*J_bar13*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 76] = 0.166666666666667*E*J_bar12*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar22*det_J*nu / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 77] = 0.166666666666667*E*J_bar12*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 78] = 0.166666666666667*E*J_bar13*J_bar13 * det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar23 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_bar33 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 79] = 0.166666666666667*E*J_bar13*J_bar23*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 80] = 0.166666666666667*E*J_bar13*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 81] = 0.166666666666667*E*J_bar13*J_star1*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 82] = 0.166666666666667*E*J_bar13*J_star2*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 83] = 0.166666666666667*E*J_bar13*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 84] = 0.166666666666667*E*J_bar11*J_bar23*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar21*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 85] = 0.166666666666667*E*J_bar11*J_bar13*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar23*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 86] = 0.166666666666667*E*J_bar21*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 87] = 0.166666666666667*E*J_bar12*J_bar23*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar22*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 88] = 0.166666666666667*E*J_bar12*J_bar13*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar23*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 89] = 0.166666666666667*E*J_bar22*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 90] = 0.166666666666667*E*J_bar13*J_bar23*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 91] = 0.166666666666667*E*J_bar13*J_bar13 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar23 * det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_bar33 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 92] = 0.166666666666667*E*J_bar23*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 93] = 0.166666666666667*E*J_bar13*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_star1*det_J*nu / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 94] = 0.166666666666667*E*J_bar13*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_star2*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 95] = 0.166666666666667*E*J_bar23*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 96] = 0.166666666666667*E*J_bar11*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 97] = 0.166666666666667*E*J_bar21*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 98] = 0.166666666666667*E*J_bar11*J_bar13*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar33*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 99] = 0.166666666666667*E*J_bar12*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 100] = 0.166666666666667*E*J_bar22*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 101] = 0.166666666666667*E*J_bar12*J_bar13*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_bar33*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 102] = 0.166666666666667*E*J_bar13*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 103] = 0.166666666666667*E*J_bar23*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 104] = 0.166666666666667*E*J_bar13*J_bar13 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar23 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_bar33 * det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 105] = 0.166666666666667*E*J_bar13*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star1*det_J*nu / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 106] = 0.166666666666667*E*J_bar23*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star2*det_J*nu / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 107] = 0.166666666666667*E*J_bar13*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star3*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 108] = 0.166666666666667*E*J_bar11*J_star1*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 109] = 0.166666666666667*E*J_bar11*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_star1*det_J*nu / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 110] = 0.166666666666667*E*J_bar11*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star1*det_J*nu / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 111] = 0.166666666666667*E*J_bar12*J_star1*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 112] = 0.166666666666667*E*J_bar12*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_star1*det_J*nu / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 113] = 0.166666666666667*E*J_bar12*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star1*det_J*nu / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 114] = 0.166666666666667*E*J_bar13*J_star1*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 115] = 0.166666666666667*E*J_bar13*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_star1*det_J*nu / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 116] = 0.166666666666667*E*J_bar13*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star1*det_J*nu / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 117] = 0.166666666666667*E*J_star1*J_star1 * det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star2*J_star2 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star3*J_star3 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 118] = 0.166666666666667*E*J_star1*J_star2*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star1*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 119] = 0.166666666666667*E*J_star1*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star1*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 120] = 0.166666666666667*E*J_bar11*J_star2*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 121] = 0.166666666666667*E*J_bar11*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_star2*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 122] = 0.166666666666667*E*J_bar21*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star2*det_J*nu / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 123] = 0.166666666666667*E*J_bar12*J_star2*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 124] = 0.166666666666667*E*J_bar12*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_star2*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 125] = 0.166666666666667*E*J_bar22*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star2*det_J*nu / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 126] = 0.166666666666667*E*J_bar13*J_star2*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 127] = 0.166666666666667*E*J_bar13*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_star2*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 128] = 0.166666666666667*E*J_bar23*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star2*det_J*nu / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 129] = 0.166666666666667*E*J_star1*J_star2*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star1*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 130] = 0.166666666666667*E*J_star1*J_star1 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star2*J_star2 * det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star3*J_star3 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 131] = 0.166666666666667*E*J_star2*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star2*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 132] = 0.166666666666667*E*J_bar11*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 133] = 0.166666666666667*E*J_bar21*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 134] = 0.166666666666667*E*J_bar11*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star3*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 135] = 0.166666666666667*E*J_bar12*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 136] = 0.166666666666667*E*J_bar22*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 137] = 0.166666666666667*E*J_bar12*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star3*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 138] = 0.166666666666667*E*J_bar13*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 139] = 0.166666666666667*E*J_bar23*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 140] = 0.166666666666667*E*J_bar13*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star3*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 141] = 0.166666666666667*E*J_star1*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star1*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 142] = 0.166666666666667*E*J_star2*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star2*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
		E_vector[offset*144 + 143] = 0.166666666666667*E*J_star1*J_star1 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star2*J_star2 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star3*J_star3 * det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1);
		

		counter = 0;
		for (int npe = 0; npe < 4; npe++){
			dummy_node = nodesInElem[nodesinelemX(npe, offset, 4)]; // The row of the matrix we looking at will be k_th element and npe (nodes per element) 	
			for (int dof = 0; dof < 3; dof++){

				DOF[counter] = displaceInElem_device[nodesDisplacementX(dof, dummy_node, 3)];
				counter++;
			}
		}

		//we will use atomic add because we will be writting to a single location multiple times (perhaps) 
		for (int c = 0; c < 12; c++){
			for (int r = 0; r < 12; r++){

				//d_A_dense[IDX2C(DOF[c], DOF[r], 3000)] = d_A_dense[IDX2C(DOF[c], DOF[r], 3000)] + E_vector[offset * 144 + c*12+r];
				atomicAdda(&(d_A_dense[IDX2C(DOF[c], DOF[r], 3 * numnodes)]), E_vector[offset * 144 + c * 12 + r]);
				//IDX2C(DOF[c], DOF[r], 3000)
				//K[IDX2C(DOF[r], DOF[c], numP*dim)] = K[IDX2C(DOF[r], DOF[c], numP*dim)] + E[k][r][c];
			}
		}


}

//This is for the global K matrix
__global__ void make_global_K(){

}

//working version??
//__global__ void make_K_cuda(double *E_vector, int *nodesInElem, double *x_vector, double *y_vector, double *z_vector) {
//	int x = threadIdx.x + blockIdx.x*blockDim.x; //if we have a 3D problem then this will go from 0 to 11
//	int y = threadIdx.y + blockIdx.y*blockDim.y; //the blockdim for x and y should be the same, which is 12
//	int z = threadIdx.z + blockIdx.z*blockDim.z; //This will control 
//	int offset = x + 12 * (y + z * 12);
//	int max_limit = 12 * 12*4374;
//	double E = 200000;
//	double nu = 0.45;
//	double x14 = x_vector[nodesInElem[nodesinelemX(0, z, 4)]] - x_vector[nodesInElem[nodesinelemX(3, z, 4)]];
//	double x24 = x_vector[nodesInElem[nodesinelemX(1, z, 4)]] - x_vector[nodesInElem[nodesinelemX(3, z, 4)]];
//	double x34 = x_vector[nodesInElem[nodesinelemX(2, z, 4)]] - x_vector[nodesInElem[nodesinelemX(3, z, 4)]];
//	double y14 = y_vector[nodesInElem[nodesinelemX(0, z, 4)]] - y_vector[nodesInElem[nodesinelemX(3, z, 4)]];
//	double y24 = y_vector[nodesInElem[nodesinelemX(1, z, 4)]] - y_vector[nodesInElem[nodesinelemX(3, z, 4)]];
//	double y34 = y_vector[nodesInElem[nodesinelemX(2, z, 4)]] - y_vector[nodesInElem[nodesinelemX(3, z, 4)]];
//	double z14 = z_vector[nodesInElem[nodesinelemX(0, z, 4)]] - z_vector[nodesInElem[nodesinelemX(3, z, 4)]];
//	double z24 = z_vector[nodesInElem[nodesinelemX(1, z, 4)]] - z_vector[nodesInElem[nodesinelemX(3, z, 4)]];
//	double z34 = z_vector[nodesInElem[nodesinelemX(2, z, 4)]] - z_vector[nodesInElem[nodesinelemX(3, z, 4)]];
//
//	//std::cout << x14*(y24*z34 - y34*z24) - y14*(x24*z34 - z24 * 34) + z14*(x24*y34 - y24*x34) << std::endl;
//	double det_J = (x14*(y24*z34 - y34*z24) - y14*(x24*z34 - z24 * x34) + z14*(x24*y34 - y24*x34));
//
//	double J_bar11 = (y24*z34 - z24*y34) / det_J;
//	double J_bar12 = (z14*y34 - y14*z34) / det_J;
//	double J_bar13 = (y14*z24 - z14*y24) / det_J;
//	double J_bar21 = (z24*x34 - x24*z34) / det_J;
//	double J_bar22 = (x14*z34 - z14*x34) / det_J;
//	double J_bar23 = (z14*x24 - x14*z24) / det_J;
//	double J_bar31 = (x24*y34 - y24*x34) / det_J;
//	double J_bar32 = (y14*x34 - x14*y34) / det_J;
//	double J_bar33 = (x14*y24 - y14*x24) / det_J;
//
//	double J_star1 = -(J_bar11 + J_bar12 + J_bar13);
//	double J_star2 = -(J_bar21 + J_bar22 + J_bar23);
//	double J_star3 = -(J_bar31 + J_bar32 + J_bar33);
//	//__syncthreads();
//	//B_Matrix testing
//	//if ((x == 0) && (y == 0)){ E_vector[offset] = J_bar11; }
//	//if ((x == 0) && (y == 1)){ E_vector[offset] = 0; }
//	//	if ((x == 0) && (y == 2)){ E_vector[offset] = 0; }
//	//	if ((x == 0) && (y == 3)){ E_vector[offset] = J_bar12; }
//	//	if ((x == 0) && (y == 4)){ E_vector[offset] = 0; }
//	//	if ((x == 0) && (y == 5)){ E_vector[offset] = 0; }
//	//	if ((x == 0) && (y == 6)){ E_vector[offset] = J_bar13; }
//	//	if ((x == 0) && (y == 7)){ E_vector[offset] = 0; }
//	//	if ((x == 0) && (y == 8)){ E_vector[offset] = 0; }
//	//	if ((x == 0) && (y == 9)){ E_vector[offset] = J_star1; }
//	//	if ((x == 0) && (y == 10)){ E_vector[offset] = 0; }
//	//	if ((x == 0) && (y == 11)){ E_vector[offset] = 0; }
//	//	if ((x == 1) && (y == 0)){ E_vector[offset] = 0; }
//	//	if ((x == 1) && (y == 1)){ E_vector[offset] = J_bar21; }
//	//	if ((x == 1) && (y == 2)){ E_vector[offset] = 0; }
//	//	if ((x == 1) && (y == 3)){ E_vector[offset] = 0; }
//	//	if ((x == 1) && (y == 4)){ E_vector[offset] = J_bar22; }
//	//	if ((x == 1) && (y == 5)){ E_vector[offset] = 0; }
//	//	if ((x == 1) && (y == 6)){ E_vector[offset] = 0; }
//	//	if ((x == 1) && (y == 7)){ E_vector[offset] = J_bar23; }
//	//	if ((x == 1) && (y == 8)){ E_vector[offset] = 0; }
//	//	if ((x == 1) && (y == 9)){ E_vector[offset] = 0; }
//	//	if ((x == 1) && (y == 10)){ E_vector[offset] = J_star2; }
//	//	if ((x == 1) && (y == 11)){ E_vector[offset] = 0; }
//	//	if ((x == 2) && (y == 0)){ E_vector[offset] = 0; }
//	//	if ((x == 2) && (y == 1)){ E_vector[offset] = 0; }
//	//	if ((x == 2) && (y == 2)){ E_vector[offset] = J_bar31; }
//	//	if ((x == 2) && (y == 3)){ E_vector[offset] = 0; }
//	//	if ((x == 2) && (y == 4)){ E_vector[offset] = 0; }
//	//	if ((x == 2) && (y == 5)){ E_vector[offset] = J_bar32; }
//	//	if ((x == 2) && (y == 6)){ E_vector[offset] = 0; }
//	//	if ((x == 2) && (y == 7)){ E_vector[offset] = 0; }
//	//	if ((x == 2) && (y == 8)){ E_vector[offset] = J_bar33; }
//	//	if ((x == 2) && (y == 9)){ E_vector[offset] = 0; }
//	//	if ((x == 2) && (y == 10)){ E_vector[offset] = 0; }
//	//	if ((x == 2) && (y == 11)){ E_vector[offset] = J_star3; }
//	//	if ((x == 3) && (y == 0)){ E_vector[offset] = J_bar21; }
//	//	if ((x == 3) && (y == 1)){ E_vector[offset] = J_bar11; }
//	//	if ((x == 3) && (y == 2)){ E_vector[offset] = 0; }
//	//	if ((x == 3) && (y == 3)){ E_vector[offset] = J_bar22; }
//	//	if ((x == 3) && (y == 4)){ E_vector[offset] = J_bar12; }
//	//	if ((x == 3) && (y == 5)){ E_vector[offset] = 0; }
//	//	if ((x == 3) && (y == 6)){ E_vector[offset] = J_bar23; }
//	//	if ((x == 3) && (y == 7)){ E_vector[offset] = J_bar13; }
//	//	if ((x == 3) && (y == 8)){ E_vector[offset] = 0; }
//	//	if ((x == 3) && (y == 9)){ E_vector[offset] = J_star2; }
//	//	if ((x == 3) && (y == 10)){ E_vector[offset] = J_star1; }
//	//	if ((x == 3) && (y == 11)){ E_vector[offset] = 0; }
//	//	if ((x == 4) && (y == 0)){ E_vector[offset] = 0; }
//	//	if ((x == 4) && (y == 1)){ E_vector[offset] = J_bar31; }
//	//	if ((x == 4) && (y == 2)){ E_vector[offset] = J_bar21; }
//	//	if ((x == 4) && (y == 3)){ E_vector[offset] = 0; }
//	//	if ((x == 4) && (y == 4)){ E_vector[offset] = J_bar32; }
//	//	if ((x == 4) && (y == 5)){ E_vector[offset] = J_bar22; }
//	//	if ((x == 4) && (y == 6)){ E_vector[offset] = 0; }
//	//	if ((x == 4) && (y == 7)){ E_vector[offset] = J_bar33; }
//	//	if ((x == 4) && (y == 8)){ E_vector[offset] = J_bar23; }
//	//	if ((x == 4) && (y == 9)){ E_vector[offset] = 0; }
//	//	if ((x == 4) && (y == 10)){ E_vector[offset] = J_star3; }
//	//	if ((x == 4) && (y == 11)){ E_vector[offset] = J_star2; }
//	//	if ((x == 5) && (y == 0)){ E_vector[offset] = J_bar31; }
//	//	if ((x == 5) && (y == 1)){ E_vector[offset] = 0; }
//	//	if ((x == 5) && (y == 2)){ E_vector[offset] = J_bar11; }
//	//	if ((x == 5) && (y == 3)){ E_vector[offset] = J_bar32; }
//	//	if ((x == 5) && (y == 4)){ E_vector[offset] = 0; }
//	//	if ((x == 5) && (y == 5)){ E_vector[offset] = J_bar12; }
//	//	if ((x == 5) && (y == 6)){ E_vector[offset] = J_bar33; }
//	//	if ((x == 5) && (y == 7)){ E_vector[offset] = 0; }
//	//	if ((x == 5) && (y == 8)){ E_vector[offset] = J_bar13; }
//	//	if ((x == 5) && (y == 9)){ E_vector[offset] = J_star3; }
//	//	if ((x == 5) && (y == 10)){ E_vector[offset] = 0; }
//	//	if ((x == 5) && (y == 11)){ E_vector[offset] = J_star1; }
//	
//		if ((x == 0) && (y == 0)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar11* det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar21 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar31 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 0) && (y == 1)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar21*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar11*J_bar21*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 0) && (y == 2)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar11*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 0) && (y == 3)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar12*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar22*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 0) && (y == 4)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar22*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar21*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 0) && (y == 5)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 0) && (y == 6)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar13*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 0) && (y == 7)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar23*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar21*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 0) && (y == 8)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 0) && (y == 9)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_star1*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 0) && (y == 10)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_star2*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 0) && (y == 11)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 1) && (y == 0)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar21*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar11*J_bar21*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 1) && (y == 1)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar11* det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar21 * det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar31 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 1) && (y == 2)){ E_vector[offset] = 0.166666666666667*E*J_bar21*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 1) && (y == 3)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar22*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar21*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 1) && (y == 4)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar12*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar22*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 1) && (y == 5)){ E_vector[offset] = 0.166666666666667*E*J_bar21*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 1) && (y == 6)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar21*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 1) && (y == 7)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar13*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar23*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 1) && (y == 8)){ E_vector[offset] = 0.166666666666667*E*J_bar21*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 1) && (y == 9)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_star1*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 1) && (y == 10)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_star2*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 1) && (y == 11)){ E_vector[offset] = 0.166666666666667*E*J_bar21*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 2) && (y == 0)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar11*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 2) && (y == 1)){ E_vector[offset] = 0.166666666666667*E*J_bar21*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 2) && (y == 2)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar11* det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar21 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar31 * det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1); }
//		if ((x == 2) && (y == 3)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 2) && (y == 4)){ E_vector[offset] = 0.166666666666667*E*J_bar21*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 2) && (y == 5)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar12*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar22*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar32*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1); }
//		if ((x == 2) && (y == 6)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 2) && (y == 7)){ E_vector[offset] = 0.166666666666667*E*J_bar21*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 2) && (y == 8)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar13*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar33*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1); }
//		if ((x == 2) && (y == 9)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star1*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 2) && (y == 10)){ E_vector[offset] = 0.166666666666667*E*J_bar21*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star2*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 2) && (y == 11)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star3*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1); }
//		if ((x == 3) && (y == 0)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar12*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar22*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 3) && (y == 1)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar22*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar21*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 3) && (y == 2)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 3) && (y == 3)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_bar12* det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar22 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_bar32 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 3) && (y == 4)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_bar22*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar22*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 3) && (y == 5)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 3) && (y == 6)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_bar13*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 3) && (y == 7)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_bar23*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar22*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 3) && (y == 8)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 3) && (y == 9)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_star1*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 3) && (y == 10)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_star2*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 3) && (y == 11)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 4) && (y == 0)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar22*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar21*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 4) && (y == 1)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar12*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar22*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 4) && (y == 2)){ E_vector[offset] = 0.166666666666667*E*J_bar21*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 4) && (y == 3)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_bar22*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar22*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 4) && (y == 4)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_bar12* det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar22 * det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_bar32 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 4) && (y == 5)){ E_vector[offset] = 0.166666666666667*E*J_bar22*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 4) && (y == 6)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar22*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 4) && (y == 7)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_bar13*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar23*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 4) && (y == 8)){ E_vector[offset] = 0.166666666666667*E*J_bar22*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 4) && (y == 9)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_star1*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 4) && (y == 10)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_star2*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 4) && (y == 11)){ E_vector[offset] = 0.166666666666667*E*J_bar22*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 5) && (y == 0)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 5) && (y == 1)){ E_vector[offset] = 0.166666666666667*E*J_bar21*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 5) && (y == 2)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar12*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar22*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar32*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1); }
//		if ((x == 5) && (y == 3)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 5) && (y == 4)){ E_vector[offset] = 0.166666666666667*E*J_bar22*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 5) && (y == 5)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_bar12* det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar22 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_bar32 * det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1); }
//		if ((x == 5) && (y == 6)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 5) && (y == 7)){ E_vector[offset] = 0.166666666666667*E*J_bar22*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 5) && (y == 8)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_bar13*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_bar33*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1); }
//		if ((x == 5) && (y == 9)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star1*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 5) && (y == 10)){ E_vector[offset] = 0.166666666666667*E*J_bar22*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star2*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 5) && (y == 11)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star3*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1); }
//		if ((x == 6) && (y == 0)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar13*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 6) && (y == 1)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar21*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 6) && (y == 2)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 6) && (y == 3)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_bar13*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 6) && (y == 4)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar22*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 6) && (y == 5)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 6) && (y == 6)){ E_vector[offset] = 0.166666666666667*E*J_bar13*J_bar13 * det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar23 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_bar33 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 6) && (y == 7)){ E_vector[offset] = 0.166666666666667*E*J_bar13*J_bar23*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 6) && (y == 8)){ E_vector[offset] = 0.166666666666667*E*J_bar13*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 6) && (y == 9)){ E_vector[offset] = 0.166666666666667*E*J_bar13*J_star1*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 6) && (y == 10)){ E_vector[offset] = 0.166666666666667*E*J_bar13*J_star2*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 6) && (y == 11)){ E_vector[offset] = 0.166666666666667*E*J_bar13*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 7) && (y == 0)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar23*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar21*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 7) && (y == 1)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar13*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar23*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 7) && (y == 2)){ E_vector[offset] = 0.166666666666667*E*J_bar21*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 7) && (y == 3)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_bar23*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar22*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 7) && (y == 4)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_bar13*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar23*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 7) && (y == 5)){ E_vector[offset] = 0.166666666666667*E*J_bar22*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 7) && (y == 6)){ E_vector[offset] = 0.166666666666667*E*J_bar13*J_bar23*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 7) && (y == 7)){ E_vector[offset] = 0.166666666666667*E*J_bar13*J_bar13 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar23 * det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_bar33 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 7) && (y == 8)){ E_vector[offset] = 0.166666666666667*E*J_bar23*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 7) && (y == 9)){ E_vector[offset] = 0.166666666666667*E*J_bar13*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_star1*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 7) && (y == 10)){ E_vector[offset] = 0.166666666666667*E*J_bar13*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_star2*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 7) && (y == 11)){ E_vector[offset] = 0.166666666666667*E*J_bar23*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 8) && (y == 0)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 8) && (y == 1)){ E_vector[offset] = 0.166666666666667*E*J_bar21*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 8) && (y == 2)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar13*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar33*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1); }
//		if ((x == 8) && (y == 3)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 8) && (y == 4)){ E_vector[offset] = 0.166666666666667*E*J_bar22*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 8) && (y == 5)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_bar13*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_bar33*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1); }
//		if ((x == 8) && (y == 6)){ E_vector[offset] = 0.166666666666667*E*J_bar13*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 8) && (y == 7)){ E_vector[offset] = 0.166666666666667*E*J_bar23*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 8) && (y == 8)){ E_vector[offset] = 0.166666666666667*E*J_bar13*J_bar13 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar23 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_bar33 * det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1); }
//		if ((x == 8) && (y == 9)){ E_vector[offset] = 0.166666666666667*E*J_bar13*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star1*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 8) && (y == 10)){ E_vector[offset] = 0.166666666666667*E*J_bar23*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star2*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 8) && (y == 11)){ E_vector[offset] = 0.166666666666667*E*J_bar13*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star3*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1); }
//		if ((x == 9) && (y == 0)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_star1*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 9) && (y == 1)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_star1*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 9) && (y == 2)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star1*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 9) && (y == 3)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_star1*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 9) && (y == 4)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_star1*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 9) && (y == 5)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star1*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 9) && (y == 6)){ E_vector[offset] = 0.166666666666667*E*J_bar13*J_star1*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 9) && (y == 7)){ E_vector[offset] = 0.166666666666667*E*J_bar13*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_star1*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 9) && (y == 8)){ E_vector[offset] = 0.166666666666667*E*J_bar13*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star1*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 9) && (y == 9)){ E_vector[offset] = 0.166666666666667*E*J_star1*J_star1 * det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star2*J_star2 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star3*J_star3 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 9) && (y == 10)){ E_vector[offset] = 0.166666666666667*E*J_star1*J_star2*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star1*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 9) && (y == 11)){ E_vector[offset] = 0.166666666666667*E*J_star1*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star1*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 10) && (y == 0)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_star2*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 10) && (y == 1)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_star2*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 10) && (y == 2)){ E_vector[offset] = 0.166666666666667*E*J_bar21*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star2*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 10) && (y == 3)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_star2*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 10) && (y == 4)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_star2*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 10) && (y == 5)){ E_vector[offset] = 0.166666666666667*E*J_bar22*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star2*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 10) && (y == 6)){ E_vector[offset] = 0.166666666666667*E*J_bar13*J_star2*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 10) && (y == 7)){ E_vector[offset] = 0.166666666666667*E*J_bar13*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_star2*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 10) && (y == 8)){ E_vector[offset] = 0.166666666666667*E*J_bar23*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star2*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 10) && (y == 9)){ E_vector[offset] = 0.166666666666667*E*J_star1*J_star2*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star1*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 10) && (y == 10)){ E_vector[offset] = 0.166666666666667*E*J_star1*J_star1 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star2*J_star2 * det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star3*J_star3 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 10) && (y == 11)){ E_vector[offset] = 0.166666666666667*E*J_star2*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star2*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 11) && (y == 0)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 11) && (y == 1)){ E_vector[offset] = 0.166666666666667*E*J_bar21*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 11) && (y == 2)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star3*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1); }
//		if ((x == 11) && (y == 3)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 11) && (y == 4)){ E_vector[offset] = 0.166666666666667*E*J_bar22*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 11) && (y == 5)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star3*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1); }
//		if ((x == 11) && (y == 6)){ E_vector[offset] = 0.166666666666667*E*J_bar13*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 11) && (y == 7)){ E_vector[offset] = 0.166666666666667*E*J_bar23*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 11) && (y == 8)){ E_vector[offset] = 0.166666666666667*E*J_bar13*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star3*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1); }
//		if ((x == 11) && (y == 9)){ E_vector[offset] = 0.166666666666667*E*J_star1*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star1*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 11) && (y == 10)){ E_vector[offset] = 0.166666666666667*E*J_star2*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star2*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 11) && (y == 11)){ E_vector[offset] = 0.166666666666667*E*J_star1*J_star1 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star2*J_star2 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star3*J_star3 * det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1); }
//		//__syncthreads();
//	
//}
//

//
//__global__ void make_K_cuda(double *E_vector, int *nodesInElem, double *x_vector, double *y_vector, double *z_vector) {
//	int x = threadIdx.x + blockIdx.x*blockDim.x; //if we have a 3D problem then this will go from 0 to 11
//	int y = threadIdx.y + blockIdx.y*blockDim.y; //the blockdim for x and y should be the same, which is 12
//	int z = threadIdx.z + blockIdx.z*blockDim.z; //This will control 
//	int offset = x + 12 * (y + z * 12);
//	int max_limit = 12 * 12*4374;
//	double E = 200000;
//	double nu = 0.45;
//	double x14 = x_vector[nodesInElem[nodesinelemX(0, z, 4)]] - x_vector[nodesInElem[nodesinelemX(3, z, 4)]];
//	double x24 = x_vector[nodesInElem[nodesinelemX(1, z, 4)]] - x_vector[nodesInElem[nodesinelemX(3, z, 4)]];
//	double x34 = x_vector[nodesInElem[nodesinelemX(2, z, 4)]] - x_vector[nodesInElem[nodesinelemX(3, z, 4)]];
//	double y14 = y_vector[nodesInElem[nodesinelemX(0, z, 4)]] - y_vector[nodesInElem[nodesinelemX(3, z, 4)]];
//	double y24 = y_vector[nodesInElem[nodesinelemX(1, z, 4)]] - y_vector[nodesInElem[nodesinelemX(3, z, 4)]];
//	double y34 = y_vector[nodesInElem[nodesinelemX(2, z, 4)]] - y_vector[nodesInElem[nodesinelemX(3, z, 4)]];
//	double z14 = z_vector[nodesInElem[nodesinelemX(0, z, 4)]] - z_vector[nodesInElem[nodesinelemX(3, z, 4)]];
//	double z24 = z_vector[nodesInElem[nodesinelemX(1, z, 4)]] - z_vector[nodesInElem[nodesinelemX(3, z, 4)]];
//	double z34 = z_vector[nodesInElem[nodesinelemX(2, z, 4)]] - z_vector[nodesInElem[nodesinelemX(3, z, 4)]];
//
//	//std::cout << x14*(y24*z34 - y34*z24) - y14*(x24*z34 - z24 * 34) + z14*(x24*y34 - y24*x34) << std::endl;
//	double det_J = (x14*(y24*z34 - y34*z24) - y14*(x24*z34 - z24 * x34) + z14*(x24*y34 - y24*x34));
//
//	double J_bar11 = (y24*z34 - z24*y34) / det_J;
//	double J_bar12 = (z14*y34 - y14*z34) / det_J;
//	double J_bar13 = (y14*z24 - z14*y24) / det_J;
//	double J_bar21 = (z24*x34 - x24*z34) / det_J;
//	double J_bar22 = (x14*z34 - z14*x34) / det_J;
//	double J_bar23 = (z14*x24 - x14*z24) / det_J;
//	double J_bar31 = (x24*y34 - y24*x34) / det_J;
//	double J_bar32 = (y14*x34 - x14*y34) / det_J;
//	double J_bar33 = (x14*y24 - y14*x24) / det_J;
//
//	double J_star1 = -(J_bar11 + J_bar12 + J_bar13);
//	double J_star2 = -(J_bar21 + J_bar22 + J_bar23);
//	double J_star3 = -(J_bar31 + J_bar32 + J_bar33);
//	//__syncthreads();
//	//B_Matrix testing
//	//if ((x == 0) && (y == 0)){ E_vector[offset] = J_bar11; }
//	//if ((x == 0) && (y == 1)){ E_vector[offset] = 0; }
//	//	if ((x == 0) && (y == 2)){ E_vector[offset] = 0; }
//	//	if ((x == 0) && (y == 3)){ E_vector[offset] = J_bar12; }
//	//	if ((x == 0) && (y == 4)){ E_vector[offset] = 0; }
//	//	if ((x == 0) && (y == 5)){ E_vector[offset] = 0; }
//	//	if ((x == 0) && (y == 6)){ E_vector[offset] = J_bar13; }
//	//	if ((x == 0) && (y == 7)){ E_vector[offset] = 0; }
//	//	if ((x == 0) && (y == 8)){ E_vector[offset] = 0; }
//	//	if ((x == 0) && (y == 9)){ E_vector[offset] = J_star1; }
//	//	if ((x == 0) && (y == 10)){ E_vector[offset] = 0; }
//	//	if ((x == 0) && (y == 11)){ E_vector[offset] = 0; }
//	//	if ((x == 1) && (y == 0)){ E_vector[offset] = 0; }
//	//	if ((x == 1) && (y == 1)){ E_vector[offset] = J_bar21; }
//	//	if ((x == 1) && (y == 2)){ E_vector[offset] = 0; }
//	//	if ((x == 1) && (y == 3)){ E_vector[offset] = 0; }
//	//	if ((x == 1) && (y == 4)){ E_vector[offset] = J_bar22; }
//	//	if ((x == 1) && (y == 5)){ E_vector[offset] = 0; }
//	//	if ((x == 1) && (y == 6)){ E_vector[offset] = 0; }
//	//	if ((x == 1) && (y == 7)){ E_vector[offset] = J_bar23; }
//	//	if ((x == 1) && (y == 8)){ E_vector[offset] = 0; }
//	//	if ((x == 1) && (y == 9)){ E_vector[offset] = 0; }
//	//	if ((x == 1) && (y == 10)){ E_vector[offset] = J_star2; }
//	//	if ((x == 1) && (y == 11)){ E_vector[offset] = 0; }
//	//	if ((x == 2) && (y == 0)){ E_vector[offset] = 0; }
//	//	if ((x == 2) && (y == 1)){ E_vector[offset] = 0; }
//	//	if ((x == 2) && (y == 2)){ E_vector[offset] = J_bar31; }
//	//	if ((x == 2) && (y == 3)){ E_vector[offset] = 0; }
//	//	if ((x == 2) && (y == 4)){ E_vector[offset] = 0; }
//	//	if ((x == 2) && (y == 5)){ E_vector[offset] = J_bar32; }
//	//	if ((x == 2) && (y == 6)){ E_vector[offset] = 0; }
//	//	if ((x == 2) && (y == 7)){ E_vector[offset] = 0; }
//	//	if ((x == 2) && (y == 8)){ E_vector[offset] = J_bar33; }
//	//	if ((x == 2) && (y == 9)){ E_vector[offset] = 0; }
//	//	if ((x == 2) && (y == 10)){ E_vector[offset] = 0; }
//	//	if ((x == 2) && (y == 11)){ E_vector[offset] = J_star3; }
//	//	if ((x == 3) && (y == 0)){ E_vector[offset] = J_bar21; }
//	//	if ((x == 3) && (y == 1)){ E_vector[offset] = J_bar11; }
//	//	if ((x == 3) && (y == 2)){ E_vector[offset] = 0; }
//	//	if ((x == 3) && (y == 3)){ E_vector[offset] = J_bar22; }
//	//	if ((x == 3) && (y == 4)){ E_vector[offset] = J_bar12; }
//	//	if ((x == 3) && (y == 5)){ E_vector[offset] = 0; }
//	//	if ((x == 3) && (y == 6)){ E_vector[offset] = J_bar23; }
//	//	if ((x == 3) && (y == 7)){ E_vector[offset] = J_bar13; }
//	//	if ((x == 3) && (y == 8)){ E_vector[offset] = 0; }
//	//	if ((x == 3) && (y == 9)){ E_vector[offset] = J_star2; }
//	//	if ((x == 3) && (y == 10)){ E_vector[offset] = J_star1; }
//	//	if ((x == 3) && (y == 11)){ E_vector[offset] = 0; }
//	//	if ((x == 4) && (y == 0)){ E_vector[offset] = 0; }
//	//	if ((x == 4) && (y == 1)){ E_vector[offset] = J_bar31; }
//	//	if ((x == 4) && (y == 2)){ E_vector[offset] = J_bar21; }
//	//	if ((x == 4) && (y == 3)){ E_vector[offset] = 0; }
//	//	if ((x == 4) && (y == 4)){ E_vector[offset] = J_bar32; }
//	//	if ((x == 4) && (y == 5)){ E_vector[offset] = J_bar22; }
//	//	if ((x == 4) && (y == 6)){ E_vector[offset] = 0; }
//	//	if ((x == 4) && (y == 7)){ E_vector[offset] = J_bar33; }
//	//	if ((x == 4) && (y == 8)){ E_vector[offset] = J_bar23; }
//	//	if ((x == 4) && (y == 9)){ E_vector[offset] = 0; }
//	//	if ((x == 4) && (y == 10)){ E_vector[offset] = J_star3; }
//	//	if ((x == 4) && (y == 11)){ E_vector[offset] = J_star2; }
//	//	if ((x == 5) && (y == 0)){ E_vector[offset] = J_bar31; }
//	//	if ((x == 5) && (y == 1)){ E_vector[offset] = 0; }
//	//	if ((x == 5) && (y == 2)){ E_vector[offset] = J_bar11; }
//	//	if ((x == 5) && (y == 3)){ E_vector[offset] = J_bar32; }
//	//	if ((x == 5) && (y == 4)){ E_vector[offset] = 0; }
//	//	if ((x == 5) && (y == 5)){ E_vector[offset] = J_bar12; }
//	//	if ((x == 5) && (y == 6)){ E_vector[offset] = J_bar33; }
//	//	if ((x == 5) && (y == 7)){ E_vector[offset] = 0; }
//	//	if ((x == 5) && (y == 8)){ E_vector[offset] = J_bar13; }
//	//	if ((x == 5) && (y == 9)){ E_vector[offset] = J_star3; }
//	//	if ((x == 5) && (y == 10)){ E_vector[offset] = 0; }
//	//	if ((x == 5) && (y == 11)){ E_vector[offset] = J_star1; }
//	
//		if ((x == 0) && (y == 0)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar11* det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar21 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar31 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 0) && (y == 1)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar21*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar11*J_bar21*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 0) && (y == 2)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar11*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 0) && (y == 3)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar12*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar22*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 0) && (y == 4)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar22*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar21*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 0) && (y == 5)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 0) && (y == 6)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar13*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 0) && (y == 7)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar23*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar21*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 0) && (y == 8)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 0) && (y == 9)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_star1*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 0) && (y == 10)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_star2*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 0) && (y == 11)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 1) && (y == 0)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar21*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar11*J_bar21*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 1) && (y == 1)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar11* det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar21 * det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar31 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 1) && (y == 2)){ E_vector[offset] = 0.166666666666667*E*J_bar21*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 1) && (y == 3)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar22*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar21*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 1) && (y == 4)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar12*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar22*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 1) && (y == 5)){ E_vector[offset] = 0.166666666666667*E*J_bar21*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 1) && (y == 6)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar21*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 1) && (y == 7)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar13*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar23*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 1) && (y == 8)){ E_vector[offset] = 0.166666666666667*E*J_bar21*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 1) && (y == 9)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_star1*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 1) && (y == 10)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_star2*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 1) && (y == 11)){ E_vector[offset] = 0.166666666666667*E*J_bar21*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 2) && (y == 0)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar11*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 2) && (y == 1)){ E_vector[offset] = 0.166666666666667*E*J_bar21*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 2) && (y == 2)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar11* det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar21 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar31 * det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1); }
//		if ((x == 2) && (y == 3)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 2) && (y == 4)){ E_vector[offset] = 0.166666666666667*E*J_bar21*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 2) && (y == 5)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar12*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar22*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar32*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1); }
//		if ((x == 2) && (y == 6)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 2) && (y == 7)){ E_vector[offset] = 0.166666666666667*E*J_bar21*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 2) && (y == 8)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar13*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar33*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1); }
//		if ((x == 2) && (y == 9)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star1*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 2) && (y == 10)){ E_vector[offset] = 0.166666666666667*E*J_bar21*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star2*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 2) && (y == 11)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star3*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1); }
//		if ((x == 3) && (y == 0)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar12*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar22*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 3) && (y == 1)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar22*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar21*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 3) && (y == 2)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 3) && (y == 3)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_bar12* det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar22 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_bar32 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 3) && (y == 4)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_bar22*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar22*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 3) && (y == 5)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 3) && (y == 6)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_bar13*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 3) && (y == 7)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_bar23*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar22*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 3) && (y == 8)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 3) && (y == 9)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_star1*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 3) && (y == 10)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_star2*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 3) && (y == 11)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 4) && (y == 0)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar22*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar21*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 4) && (y == 1)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar12*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar22*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 4) && (y == 2)){ E_vector[offset] = 0.166666666666667*E*J_bar21*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 4) && (y == 3)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_bar22*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar22*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 4) && (y == 4)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_bar12* det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar22 * det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_bar32 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 4) && (y == 5)){ E_vector[offset] = 0.166666666666667*E*J_bar22*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 4) && (y == 6)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar22*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 4) && (y == 7)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_bar13*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar23*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 4) && (y == 8)){ E_vector[offset] = 0.166666666666667*E*J_bar22*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 4) && (y == 9)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_star1*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 4) && (y == 10)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_star2*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 4) && (y == 11)){ E_vector[offset] = 0.166666666666667*E*J_bar22*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 5) && (y == 0)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 5) && (y == 1)){ E_vector[offset] = 0.166666666666667*E*J_bar21*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 5) && (y == 2)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar12*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar22*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar32*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1); }
//		if ((x == 5) && (y == 3)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 5) && (y == 4)){ E_vector[offset] = 0.166666666666667*E*J_bar22*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 5) && (y == 5)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_bar12* det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar22 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_bar32 * det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1); }
//		if ((x == 5) && (y == 6)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 5) && (y == 7)){ E_vector[offset] = 0.166666666666667*E*J_bar22*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 5) && (y == 8)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_bar13*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_bar33*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1); }
//		if ((x == 5) && (y == 9)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star1*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 5) && (y == 10)){ E_vector[offset] = 0.166666666666667*E*J_bar22*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star2*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 5) && (y == 11)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star3*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1); }
//		if ((x == 6) && (y == 0)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar13*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 6) && (y == 1)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar21*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 6) && (y == 2)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 6) && (y == 3)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_bar13*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 6) && (y == 4)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar22*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 6) && (y == 5)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 6) && (y == 6)){ E_vector[offset] = 0.166666666666667*E*J_bar13*J_bar13 * det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar23 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_bar33 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 6) && (y == 7)){ E_vector[offset] = 0.166666666666667*E*J_bar13*J_bar23*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 6) && (y == 8)){ E_vector[offset] = 0.166666666666667*E*J_bar13*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 6) && (y == 9)){ E_vector[offset] = 0.166666666666667*E*J_bar13*J_star1*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 6) && (y == 10)){ E_vector[offset] = 0.166666666666667*E*J_bar13*J_star2*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 6) && (y == 11)){ E_vector[offset] = 0.166666666666667*E*J_bar13*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 7) && (y == 0)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar23*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar21*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 7) && (y == 1)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar13*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar23*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 7) && (y == 2)){ E_vector[offset] = 0.166666666666667*E*J_bar21*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 7) && (y == 3)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_bar23*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar22*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 7) && (y == 4)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_bar13*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar23*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 7) && (y == 5)){ E_vector[offset] = 0.166666666666667*E*J_bar22*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 7) && (y == 6)){ E_vector[offset] = 0.166666666666667*E*J_bar13*J_bar23*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 7) && (y == 7)){ E_vector[offset] = 0.166666666666667*E*J_bar13*J_bar13 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar23 * det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_bar33 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 7) && (y == 8)){ E_vector[offset] = 0.166666666666667*E*J_bar23*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 7) && (y == 9)){ E_vector[offset] = 0.166666666666667*E*J_bar13*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_star1*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 7) && (y == 10)){ E_vector[offset] = 0.166666666666667*E*J_bar13*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_star2*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 7) && (y == 11)){ E_vector[offset] = 0.166666666666667*E*J_bar23*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 8) && (y == 0)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 8) && (y == 1)){ E_vector[offset] = 0.166666666666667*E*J_bar21*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 8) && (y == 2)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar13*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar33*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1); }
//		if ((x == 8) && (y == 3)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 8) && (y == 4)){ E_vector[offset] = 0.166666666666667*E*J_bar22*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 8) && (y == 5)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_bar13*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_bar33*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1); }
//		if ((x == 8) && (y == 6)){ E_vector[offset] = 0.166666666666667*E*J_bar13*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 8) && (y == 7)){ E_vector[offset] = 0.166666666666667*E*J_bar23*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 8) && (y == 8)){ E_vector[offset] = 0.166666666666667*E*J_bar13*J_bar13 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar23 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_bar33 * det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1); }
//		if ((x == 8) && (y == 9)){ E_vector[offset] = 0.166666666666667*E*J_bar13*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star1*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 8) && (y == 10)){ E_vector[offset] = 0.166666666666667*E*J_bar23*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star2*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 8) && (y == 11)){ E_vector[offset] = 0.166666666666667*E*J_bar13*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star3*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1); }
//		if ((x == 9) && (y == 0)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_star1*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 9) && (y == 1)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_star1*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 9) && (y == 2)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star1*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 9) && (y == 3)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_star1*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 9) && (y == 4)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_star1*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 9) && (y == 5)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star1*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 9) && (y == 6)){ E_vector[offset] = 0.166666666666667*E*J_bar13*J_star1*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 9) && (y == 7)){ E_vector[offset] = 0.166666666666667*E*J_bar13*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_star1*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 9) && (y == 8)){ E_vector[offset] = 0.166666666666667*E*J_bar13*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star1*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 9) && (y == 9)){ E_vector[offset] = 0.166666666666667*E*J_star1*J_star1 * det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star2*J_star2 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star3*J_star3 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 9) && (y == 10)){ E_vector[offset] = 0.166666666666667*E*J_star1*J_star2*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star1*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 9) && (y == 11)){ E_vector[offset] = 0.166666666666667*E*J_star1*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star1*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 10) && (y == 0)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_star2*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 10) && (y == 1)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_star2*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 10) && (y == 2)){ E_vector[offset] = 0.166666666666667*E*J_bar21*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star2*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 10) && (y == 3)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_star2*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 10) && (y == 4)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_star2*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 10) && (y == 5)){ E_vector[offset] = 0.166666666666667*E*J_bar22*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star2*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 10) && (y == 6)){ E_vector[offset] = 0.166666666666667*E*J_bar13*J_star2*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 10) && (y == 7)){ E_vector[offset] = 0.166666666666667*E*J_bar13*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_star2*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 10) && (y == 8)){ E_vector[offset] = 0.166666666666667*E*J_bar23*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star2*det_J*nu / (-2 * nu*nu - nu + 1); }
//		if ((x == 10) && (y == 9)){ E_vector[offset] = 0.166666666666667*E*J_star1*J_star2*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star1*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 10) && (y == 10)){ E_vector[offset] = 0.166666666666667*E*J_star1*J_star1 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star2*J_star2 * det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star3*J_star3 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 10) && (y == 11)){ E_vector[offset] = 0.166666666666667*E*J_star2*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star2*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 11) && (y == 0)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 11) && (y == 1)){ E_vector[offset] = 0.166666666666667*E*J_bar21*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 11) && (y == 2)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star3*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1); }
//		if ((x == 11) && (y == 3)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 11) && (y == 4)){ E_vector[offset] = 0.166666666666667*E*J_bar22*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 11) && (y == 5)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star3*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1); }
//		if ((x == 11) && (y == 6)){ E_vector[offset] = 0.166666666666667*E*J_bar13*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 11) && (y == 7)){ E_vector[offset] = 0.166666666666667*E*J_bar23*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 11) && (y == 8)){ E_vector[offset] = 0.166666666666667*E*J_bar13*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star3*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1); }
//		if ((x == 11) && (y == 9)){ E_vector[offset] = 0.166666666666667*E*J_star1*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star1*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 11) && (y == 10)){ E_vector[offset] = 0.166666666666667*E*J_star2*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star2*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
//		if ((x == 11) && (y == 11)){ E_vector[offset] = 0.166666666666667*E*J_star1*J_star1 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star2*J_star2 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star3*J_star3 * det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1); }
//		//__syncthreads();
//	
//}
////


//working 2d cuda
/*
__global__ void make_K_cuda(double *E_vector, int *nodesInElem, double *x_vector, double *y_vector, double *z_vector) {
	 __shared__ double det_J_shared[72];
	 double det_J;
	 double J_bar11;
	double J_bar12;
	double J_bar13;
	double J_bar21;
	double J_bar22;
	double J_bar23;
	double J_bar31;
	double J_bar32;
	double J_bar33 ;
	 double x14;
	 double x24;
	 double x34;
	 double y14;
	 double y24;
	 double y34;
	 double z14;
	 double z24;
	 double z34;
	double E = 200000;
	double nu = 0.45;
	int x = threadIdx.x + blockIdx.x*blockDim.x; //if we have a 3D problem then this will go from 0 to 11
	int z = threadIdx.y + blockIdx.y*blockDim.y; //the blockdim for x and y should be the same, which is 12

	int offset = x + 12 * ( z * 12);
	int max_limit = 12 * 12 * 4374;
	int cacheIndex = threadIdx.x;
	
	 x14 = x_vector[nodesInElem[nodesinelemX(0, z, 4)]] - x_vector[nodesInElem[nodesinelemX(3, z, 4)]];
	 x24 = x_vector[nodesInElem[nodesinelemX(1, z, 4)]] - x_vector[nodesInElem[nodesinelemX(3, z, 4)]];
	 x34 = x_vector[nodesInElem[nodesinelemX(2, z, 4)]] - x_vector[nodesInElem[nodesinelemX(3, z, 4)]];
	 y14 = y_vector[nodesInElem[nodesinelemX(0, z, 4)]] - y_vector[nodesInElem[nodesinelemX(3, z, 4)]];
	 y24 = y_vector[nodesInElem[nodesinelemX(1, z, 4)]] - y_vector[nodesInElem[nodesinelemX(3, z, 4)]];
	 y34 = y_vector[nodesInElem[nodesinelemX(2, z, 4)]] - y_vector[nodesInElem[nodesinelemX(3, z, 4)]];
	 z14 = z_vector[nodesInElem[nodesinelemX(0, z, 4)]] - z_vector[nodesInElem[nodesinelemX(3, z, 4)]];
	 z24 = z_vector[nodesInElem[nodesinelemX(1, z, 4)]] - z_vector[nodesInElem[nodesinelemX(3, z, 4)]];
	 z34 = z_vector[nodesInElem[nodesinelemX(2, z, 4)]] - z_vector[nodesInElem[nodesinelemX(3, z, 4)]];

	//std::cout << x14*(y24*z34 - y34*z24) - y14*(x24*z34 - z24 * 34) + z14*(x24*y34 - y24*x34) << std::endl;

	//these lines take up 0.02 ms -begin

		 det_J = (x14*(y24*z34 - y34*z24) - y14*(x24*z34 - z24 * x34) + z14*(x24*y34 - y24*x34));
		
	

	 
	 //det_J = det_J_shared[cacheIndex];
	J_bar11 = (y24*z34 - z24*y34) / det_J;
	 J_bar12 = (z14*y34 - y14*z34) / det_J;
	 J_bar13 = (y14*z24 - z14*y24) / det_J;
	 J_bar21 = (z24*x34 - x24*z34) / det_J;
	 J_bar22 = (x14*z34 - z14*x34) / det_J;
	 J_bar23 = (z14*x24 - x14*z24) / det_J;
	 J_bar31 = (x24*y34 - y24*x34) / det_J;
	 J_bar32 = (y14*x34 - x14*y34) / det_J;
	 J_bar33 = (x14*y24 - y14*x24) / det_J;

	double J_star1 = -(J_bar11 + J_bar12 + J_bar13);
	double J_star2 = -(J_bar21 + J_bar22 + J_bar23);
	double J_star3 = -(J_bar31 + J_bar32 + J_bar33);
	//-endd
	//__syncthreads();
	//B_Matrix testing
	//if ((x == 0) && (y == 0)){ E_vector[offset] = J_bar11; }
	//if ((x == 0) && (y == 1)){ E_vector[offset] = 0; }
	//	if ((x == 0) && (y == 2)){ E_vector[offset] = 0; }
	//	if ((x == 0) && (y == 3)){ E_vector[offset] = J_bar12; }
	//	if ((x == 0) && (y == 4)){ E_vector[offset] = 0; }
	//	if ((x == 0) && (y == 5)){ E_vector[offset] = 0; }
	//	if ((x == 0) && (y == 6)){ E_vector[offset] = J_bar13; }
	//	if ((x == 0) && (y == 7)){ E_vector[offset] = 0; }
	//	if ((x == 0) && (y == 8)){ E_vector[offset] = 0; }
	//	if ((x == 0) && (y == 9)){ E_vector[offset] = J_star1; }
	//	if ((x == 0) && (y == 10)){ E_vector[offset] = 0; }
	//	if ((x == 0) && (y == 11)){ E_vector[offset] = 0; }
	//	if ((x == 1) && (y == 0)){ E_vector[offset] = 0; }
	//	if ((x == 1) && (y == 1)){ E_vector[offset] = J_bar21; }
	//	if ((x == 1) && (y == 2)){ E_vector[offset] = 0; }
	//	if ((x == 1) && (y == 3)){ E_vector[offset] = 0; }
	//	if ((x == 1) && (y == 4)){ E_vector[offset] = J_bar22; }
	//	if ((x == 1) && (y == 5)){ E_vector[offset] = 0; }
	//	if ((x == 1) && (y == 6)){ E_vector[offset] = 0; }
	//	if ((x == 1) && (y == 7)){ E_vector[offset] = J_bar23; }
	//	if ((x == 1) && (y == 8)){ E_vector[offset] = 0; }
	//	if ((x == 1) && (y == 9)){ E_vector[offset] = 0; }
	//	if ((x == 1) && (y == 10)){ E_vector[offset] = J_star2; }
	//	if ((x == 1) && (y == 11)){ E_vector[offset] = 0; }
	//	if ((x == 2) && (y == 0)){ E_vector[offset] = 0; }
	//	if ((x == 2) && (y == 1)){ E_vector[offset] = 0; }
	//	if ((x == 2) && (y == 2)){ E_vector[offset] = J_bar31; }
	//	if ((x == 2) && (y == 3)){ E_vector[offset] = 0; }
	//	if ((x == 2) && (y == 4)){ E_vector[offset] = 0; }
	//	if ((x == 2) && (y == 5)){ E_vector[offset] = J_bar32; }
	//	if ((x == 2) && (y == 6)){ E_vector[offset] = 0; }
	//	if ((x == 2) && (y == 7)){ E_vector[offset] = 0; }
	//	if ((x == 2) && (y == 8)){ E_vector[offset] = J_bar33; }
	//	if ((x == 2) && (y == 9)){ E_vector[offset] = 0; }
	//	if ((x == 2) && (y == 10)){ E_vector[offset] = 0; }
	//	if ((x == 2) && (y == 11)){ E_vector[offset] = J_star3; }
	//	if ((x == 3) && (y == 0)){ E_vector[offset] = J_bar21; }
	//	if ((x == 3) && (y == 1)){ E_vector[offset] = J_bar11; }
	//	if ((x == 3) && (y == 2)){ E_vector[offset] = 0; }
	//	if ((x == 3) && (y == 3)){ E_vector[offset] = J_bar22; }
	//	if ((x == 3) && (y == 4)){ E_vector[offset] = J_bar12; }
	//	if ((x == 3) && (y == 5)){ E_vector[offset] = 0; }
	//	if ((x == 3) && (y == 6)){ E_vector[offset] = J_bar23; }
	//	if ((x == 3) && (y == 7)){ E_vector[offset] = J_bar13; }
	//	if ((x == 3) && (y == 8)){ E_vector[offset] = 0; }
	//	if ((x == 3) && (y == 9)){ E_vector[offset] = J_star2; }
	//	if ((x == 3) && (y == 10)){ E_vector[offset] = J_star1; }
	//	if ((x == 3) && (y == 11)){ E_vector[offset] = 0; }
	//	if ((x == 4) && (y == 0)){ E_vector[offset] = 0; }
	//	if ((x == 4) && (y == 1)){ E_vector[offset] = J_bar31; }
	//	if ((x == 4) && (y == 2)){ E_vector[offset] = J_bar21; }
	//	if ((x == 4) && (y == 3)){ E_vector[offset] = 0; }
	//	if ((x == 4) && (y == 4)){ E_vector[offset] = J_bar32; }
	//	if ((x == 4) && (y == 5)){ E_vector[offset] = J_bar22; }
	//	if ((x == 4) && (y == 6)){ E_vector[offset] = 0; }
	//	if ((x == 4) && (y == 7)){ E_vector[offset] = J_bar33; }
	//	if ((x == 4) && (y == 8)){ E_vector[offset] = J_bar23; }
	//	if ((x == 4) && (y == 9)){ E_vector[offset] = 0; }
	//	if ((x == 4) && (y == 10)){ E_vector[offset] = J_star3; }
	//	if ((x == 4) && (y == 11)){ E_vector[offset] = J_star2; }
	//	if ((x == 5) && (y == 0)){ E_vector[offset] = J_bar31; }
	//	if ((x == 5) && (y == 1)){ E_vector[offset] = 0; }
	//	if ((x == 5) && (y == 2)){ E_vector[offset] = J_bar11; }
	//	if ((x == 5) && (y == 3)){ E_vector[offset] = J_bar32; }
	//	if ((x == 5) && (y == 4)){ E_vector[offset] = 0; }
	//	if ((x == 5) && (y == 5)){ E_vector[offset] = J_bar12; }
	//	if ((x == 5) && (y == 6)){ E_vector[offset] = J_bar33; }
	//	if ((x == 5) && (y == 7)){ E_vector[offset] = 0; }
	//	if ((x == 5) && (y == 8)){ E_vector[offset] = J_bar13; }
	//	if ((x == 5) && (y == 9)){ E_vector[offset] = J_star3; }
	//	if ((x == 5) && (y == 10)){ E_vector[offset] = 0; }
	//	if ((x == 5) && (y == 11)){ E_vector[offset] = J_star1; }
	x = x +1;

	if ((x == 1)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar11 * det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar21 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar31 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 2)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar21*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar11*J_bar21*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 3)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar11*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 4)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar12*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar22*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 5)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar22*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar21*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 6)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 7)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar13*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 8)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar23*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar21*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 9)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 10)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_star1*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 11)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_star2*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 12)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 13)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar21*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar11*J_bar21*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 14)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar11 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar21 * det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar31 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 15)){ E_vector[offset] = 0.166666666666667*E*J_bar21*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 16)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar22*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar21*det_J*nu / (-2 * nu*nu - nu + 1); }
	if ((x == 17)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar12*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar22*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 18)){ E_vector[offset] = 0.166666666666667*E*J_bar21*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 19)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar21*det_J*nu / (-2 * nu*nu - nu + 1); }
	if ((x == 20)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar13*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar23*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 21)){ E_vector[offset] = 0.166666666666667*E*J_bar21*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 22)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_star1*det_J*nu / (-2 * nu*nu - nu + 1); }
	if ((x == 23)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_star2*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 24)){ E_vector[offset] = 0.166666666666667*E*J_bar21*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 25)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar11*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 26)){ E_vector[offset] = 0.166666666666667*E*J_bar21*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 27)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar11 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar21 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar31 * det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1); }
	if ((x == 28)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1); }
	if ((x == 29)){ E_vector[offset] = 0.166666666666667*E*J_bar21*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1); }
	if ((x == 30)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar12*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar22*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar32*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1); }
	if ((x == 31)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1); }
	if ((x == 32)){ E_vector[offset] = 0.166666666666667*E*J_bar21*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1); }
	if ((x == 33)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar13*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar33*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1); }
	if ((x == 34)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star1*det_J*nu / (-2 * nu*nu - nu + 1); }
	if ((x == 35)){ E_vector[offset] = 0.166666666666667*E*J_bar21*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star2*det_J*nu / (-2 * nu*nu - nu + 1); }
	if ((x == 36)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star3*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1); }
	if ((x == 37)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar12*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar22*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 38)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar22*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar21*det_J*nu / (-2 * nu*nu - nu + 1); }
	if ((x == 39)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1); }
	if ((x == 40)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_bar12 * det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar22 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_bar32 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 41)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_bar22*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar22*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 42)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 43)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_bar13*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 44)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_bar23*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar22*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 45)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 46)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_star1*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 47)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_star2*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 48)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 49)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar22*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar21*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 50)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar12*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar22*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 51)){ E_vector[offset] = 0.166666666666667*E*J_bar21*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1); }
	if ((x == 52)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_bar22*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar22*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 53)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_bar12 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar22 * det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_bar32 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 54)){ E_vector[offset] = 0.166666666666667*E*J_bar22*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 55)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar22*det_J*nu / (-2 * nu*nu - nu + 1); }
	if ((x == 56)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_bar13*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar23*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 57)){ E_vector[offset] = 0.166666666666667*E*J_bar22*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 58)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_star1*det_J*nu / (-2 * nu*nu - nu + 1); }
	if ((x == 59)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_star2*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 60)){ E_vector[offset] = 0.166666666666667*E*J_bar22*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 61)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 62)){ E_vector[offset] = 0.166666666666667*E*J_bar21*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 63)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar12*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar22*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar32*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1); }
	if ((x == 64)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 65)){ E_vector[offset] = 0.166666666666667*E*J_bar22*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 66)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_bar12 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar22 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_bar32 * det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1); }
	if ((x == 67)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1); }
	if ((x == 68)){ E_vector[offset] = 0.166666666666667*E*J_bar22*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1); }
	if ((x == 69)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_bar13*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_bar33*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1); }
	if ((x == 70)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star1*det_J*nu / (-2 * nu*nu - nu + 1); }
	if ((x == 71)){ E_vector[offset] = 0.166666666666667*E*J_bar22*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star2*det_J*nu / (-2 * nu*nu - nu + 1); }
	if ((x == 72)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star3*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1); }
	if ((x == 73)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar13*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 74)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar21*det_J*nu / (-2 * nu*nu - nu + 1); }
	if ((x == 75)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1); }
	if ((x == 76)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_bar13*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 77)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar22*det_J*nu / (-2 * nu*nu - nu + 1); }
	if ((x == 78)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1); }
	if ((x == 79)){ E_vector[offset] = 0.166666666666667*E*J_bar13*J_bar13 * det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar23 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_bar33 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 80)){ E_vector[offset] = 0.166666666666667*E*J_bar13*J_bar23*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 81)){ E_vector[offset] = 0.166666666666667*E*J_bar13*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 82)){ E_vector[offset] = 0.166666666666667*E*J_bar13*J_star1*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 83)){ E_vector[offset] = 0.166666666666667*E*J_bar13*J_star2*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 84)){ E_vector[offset] = 0.166666666666667*E*J_bar13*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 85)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar23*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar21*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 86)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar13*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar23*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 87)){ E_vector[offset] = 0.166666666666667*E*J_bar21*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1); }
	if ((x == 88)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_bar23*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar22*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 89)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_bar13*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar23*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 90)){ E_vector[offset] = 0.166666666666667*E*J_bar22*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1); }
	if ((x == 91)){ E_vector[offset] = 0.166666666666667*E*J_bar13*J_bar23*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 92)){ E_vector[offset] = 0.166666666666667*E*J_bar13*J_bar13 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar23 * det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_bar33 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 93)){ E_vector[offset] = 0.166666666666667*E*J_bar23*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 94)){ E_vector[offset] = 0.166666666666667*E*J_bar13*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_star1*det_J*nu / (-2 * nu*nu - nu + 1); }
	if ((x == 95)){ E_vector[offset] = 0.166666666666667*E*J_bar13*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_star2*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 96)){ E_vector[offset] = 0.166666666666667*E*J_bar23*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 97)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 98)){ E_vector[offset] = 0.166666666666667*E*J_bar21*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 99)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_bar13*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar33*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1); }
	if ((x == 100)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 101)){ E_vector[offset] = 0.166666666666667*E*J_bar22*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 102)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_bar13*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_bar33*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1); }
	if ((x == 103)){ E_vector[offset] = 0.166666666666667*E*J_bar13*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 104)){ E_vector[offset] = 0.166666666666667*E*J_bar23*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 105)){ E_vector[offset] = 0.166666666666667*E*J_bar13*J_bar13 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar23 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_bar33 * det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1); }
	if ((x == 106)){ E_vector[offset] = 0.166666666666667*E*J_bar13*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star1*det_J*nu / (-2 * nu*nu - nu + 1); }
	if ((x == 107)){ E_vector[offset] = 0.166666666666667*E*J_bar23*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star2*det_J*nu / (-2 * nu*nu - nu + 1); }
	if ((x == 108)){ E_vector[offset] = 0.166666666666667*E*J_bar13*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star3*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1); }
	if ((x == 109)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_star1*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 110)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_star1*det_J*nu / (-2 * nu*nu - nu + 1); }
	if ((x == 111)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star1*det_J*nu / (-2 * nu*nu - nu + 1); }
	if ((x == 112)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_star1*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 113)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_star1*det_J*nu / (-2 * nu*nu - nu + 1); }
	if ((x == 114)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star1*det_J*nu / (-2 * nu*nu - nu + 1); }
	if ((x == 115)){ E_vector[offset] = 0.166666666666667*E*J_bar13*J_star1*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 116)){ E_vector[offset] = 0.166666666666667*E*J_bar13*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_star1*det_J*nu / (-2 * nu*nu - nu + 1); }
	if ((x == 117)){ E_vector[offset] = 0.166666666666667*E*J_bar13*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star1*det_J*nu / (-2 * nu*nu - nu + 1); }
	if ((x == 118)){ E_vector[offset] = 0.166666666666667*E*J_star1*J_star1 * det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star2*J_star2 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star3*J_star3 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 119)){ E_vector[offset] = 0.166666666666667*E*J_star1*J_star2*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star1*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 120)){ E_vector[offset] = 0.166666666666667*E*J_star1*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star1*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 121)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_star2*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 122)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_star2*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 123)){ E_vector[offset] = 0.166666666666667*E*J_bar21*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star2*det_J*nu / (-2 * nu*nu - nu + 1); }
	if ((x == 124)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_star2*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 125)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_star2*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 126)){ E_vector[offset] = 0.166666666666667*E*J_bar22*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star2*det_J*nu / (-2 * nu*nu - nu + 1); }
	if ((x == 127)){ E_vector[offset] = 0.166666666666667*E*J_bar13*J_star2*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 128)){ E_vector[offset] = 0.166666666666667*E*J_bar13*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_star2*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 129)){ E_vector[offset] = 0.166666666666667*E*J_bar23*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star2*det_J*nu / (-2 * nu*nu - nu + 1); }
	if ((x == 130)){ E_vector[offset] = 0.166666666666667*E*J_star1*J_star2*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star1*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 131)){ E_vector[offset] = 0.166666666666667*E*J_star1*J_star1 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star2*J_star2 * det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star3*J_star3 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 132)){ E_vector[offset] = 0.166666666666667*E*J_star2*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star2*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 133)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 134)){ E_vector[offset] = 0.166666666666667*E*J_bar21*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 135)){ E_vector[offset] = 0.166666666666667*E*J_bar11*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star3*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1); }
	if ((x == 136)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 137)){ E_vector[offset] = 0.166666666666667*E*J_bar22*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 138)){ E_vector[offset] = 0.166666666666667*E*J_bar12*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star3*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1); }
	if ((x == 139)){ E_vector[offset] = 0.166666666666667*E*J_bar13*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 140)){ E_vector[offset] = 0.166666666666667*E*J_bar23*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 141)){ E_vector[offset] = 0.166666666666667*E*J_bar13*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star3*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1); }
	if ((x == 142)){ E_vector[offset] = 0.166666666666667*E*J_star1*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star1*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 143)){ E_vector[offset] = 0.166666666666667*E*J_star2*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star2*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1); }
	if ((x == 144)){ E_vector[offset] = 0.166666666666667*E*J_star1*J_star1 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star2*J_star2 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star3*J_star3 * det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1); }
	
}
//


*/