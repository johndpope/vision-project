#include "hip/hip_runtime.h"



#include "cudaFEM_read.cuh"
#include <fstream> 
#include <iostream>
#include <cstdio>
#include <ctime>
//#include "kinectotherfunctions.h"
//#include "kinect_vision.h"
#include "vision_main.h"

#include "FEM_draw.cuh"

using namespace std;

int main(void){


	int a;
	Geometry testing_geo;

	testing_geo.set_dim(3);
	testing_geo.read_nodes();
	testing_geo.read_elem();
	testing_geo.read_force();
	testing_geo.set_YoungPoisson(30000.0, 0.49);
	testing_geo.set_thickness(0.005);
	testing_geo.initilizeMatrices();
	testing_geo.initialize_CUDA();
	testing_geo.set_dynamic(false);
	testing_geo.set_cuda_use(true);
    //kinect_main(0, NULL,&testing_geo);
	
	 //maain(0, NULL);
	

	
	//cout << "x_value is" << testing_geo.return_y(1) << endl;
	//cout << "numNodes:" << testing_geo.return_numNodes() << endl;
	//
	//
	
	if (0){
		testing_geo.initilizeMatrices();
		std::ofstream writenodes("FEM_position_result.txt");
		double duration_K;
		double duration_solver;
		for (int i = 0; i < 20; i++){
			cout << "Iteration:" << i << endl;
			std::clock_t start_K;
			std::clock_t start_solver;

			//Assemble K
			start_K = std::clock();

			testing_geo.make_K_matrix();
			testing_geo.make_surface_f();
			

			duration_K = (std::clock() - start_K) / (double)CLOCKS_PER_SEC;

			//Solve Ax = b
			start_solver = std::clock();

			testing_geo.tt();
			duration_solver = (std::clock() - start_solver) / (double)CLOCKS_PER_SEC;
			for (int i = 0; i < testing_geo.return_numNodes(); i++) {
				writenodes << testing_geo.return_x(i) << "   " << testing_geo.return_y(i) << endl;
			}

			//cout << "K time: " << duration_K << endl;
			//cout << "Solver time: " << duration_solver << endl;
		}

		writenodes.close();
	}
 	//cuda_solver();
	//magma_solver();
	draw_things(&testing_geo);
	cin >> a; 

	return 0;
}