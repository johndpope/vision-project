#include "hip/hip_runtime.h"

#include <stdio.h>
#define _USE_MATH_DEFINES

#include <math.h>
#include <GLFW/glfw3.h>
#include <cstdlib>
#include <iostream>
#include <ctime>
#include <cstdio>
#include <linmath.h>
#include "FEM_draw.cuh"
#include "cudaFEM_read.cuh"

//Include for CMIZ
#include "zinc/context.hpp"
#include "zinc/element.hpp"
#include "zinc/field.hpp"
#include "zinc/fieldcache.hpp"
#include "zinc/fieldmodule.hpp"
#include <zinc/fieldvectoroperators.hpp>
#include "zinc/region.hpp"
#include "zinc/sceneviewer.hpp"
#include "zinc/scene.hpp"
#include <iostream>
using namespace OpenCMISS::Zinc;

#define RADIUS          150.0f
#define STEP_LONGITUDE   22.5f                   /* 22.5 makes 8 bands like original Boing */
#define STEP_LATITUDE    22.5f

#define DIST_BALL       (RADIUS * 2.f + RADIUS * 0.1f)

#define VIEW_SCENE_DIST (DIST_BALL * 3.f + 200.f)/* distance from viewer to middle of boing area */
#define GRID_SIZE       (RADIUS * 4.5f)          /* length (width) of grid */
#define BOUNCE_HEIGHT   (RADIUS * 2.1f)
#define BOUNCE_WIDTH    (RADIUS * 2.1f)

#define SHADOW_OFFSET_X -20.f
#define SHADOW_OFFSET_Y  10.f
#define SHADOW_OFFSET_Z   0.f

#define WALL_L_OFFSET   0.f
#define WALL_R_OFFSET   5.f
float3 rotate = make_float3(0.0, 0.0, 0.0);
/* Animation speed (50.0 mimics the original GLUT demo speed) */
#define ANIMATION_SPEED 50.f

/* Maximum allowed delta time per physics iteration */
#define MAX_DELTA_T 0.02f

/* Draw ball, or its shadow */
typedef enum { DRAW_BALL, DRAW_BALL_SHADOW } DRAW_BALL_ENUM;

/* Vertex type */
typedef struct { float x; float y; float z; } vertex_t;

/* Global vars */
int width, height;
GLfloat deg_rot_y = 0.f;
GLfloat deg_rot_y_inc = 2.f;
GLboolean override_pos = GL_FALSE;
GLfloat cursor_x = 0.f;
GLfloat cursor_y = 0.f;
GLfloat ball_x = -RADIUS;
GLfloat ball_y = -RADIUS;
GLfloat ball_x_inc = 1.f;
GLfloat ball_y_inc = 2.f;
DRAW_BALL_ENUM drawBallHow;
double dx, dy;
double  t;
double  t_old = 0.f;
double  dt;
double x_win_min, y_win_min;
int closest_Node = 0;
int closest_Node_new;
int node_selected;
bool changeNode = false;
float mouse_old_x = 0;
float mouse_old_y = 0;
float distance_change = 1.0f;
float3 translation = make_float3(0.0, 0.0, 0.0);
/* Random number generator */
#ifndef RAND_MAX
#define RAND_MAX 4095
#endif


/*****************************************************************************
* Truncate a degree.
*****************************************************************************/
GLfloat TruncateDeg(GLfloat deg)
{
	if (deg >= 360.f)
		return (deg - 360.f);
	else
		return deg;
}

/*****************************************************************************
* Convert a degree (360-based) into a radian.
* 360' = 2 * PI
*****************************************************************************/
double deg2rad(double deg)
{
	return deg / 360 * (2 * M_PI);
}

/*****************************************************************************
* 360' sin().
*****************************************************************************/
double sin_deg(double deg)
{
	return sin(deg2rad(deg));
}

/*****************************************************************************
* 360' cos().
*****************************************************************************/
double cos_deg(double deg)
{
	return cos(deg2rad(deg));
}

/*****************************************************************************
* Compute a cross product (for a normal vector).
*
* c = a x b
*****************************************************************************/
void CrossProduct(vertex_t a, vertex_t b, vertex_t c, vertex_t *n)
{
	GLfloat u1, u2, u3;
	GLfloat v1, v2, v3;

	u1 = b.x - a.x;
	u2 = b.y - a.y;
	u3 = b.y - a.z;

	v1 = c.x - a.x;
	v2 = c.y - a.y;
	v3 = c.z - a.z;

	n->x = u2 * v3 - v2 * v3;
	n->y = u3 * v1 - v3 * u1;
	n->z = u1 * v2 - v1 * u2;
}


#define BOING_DEBUG 0


/*****************************************************************************
* init()
*****************************************************************************/
void init(void)
{
	/*
	* Clear background.
	*/
	glClearColor(0.55f, 0.55f, 0.55f, 0.f);

	glShadeModel(GL_FLAT);
}


/*****************************************************************************
* display()
*****************************************************************************/
void display(void)
{
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
	glPushMatrix();

	drawBallHow = DRAW_BALL_SHADOW;
	DrawBoingBall();

	DrawGrid();

	drawBallHow = DRAW_BALL;
	DrawBoingBall();

	glPopMatrix();
	glFlush();
}


/*****************************************************************************
* reshape()
*****************************************************************************/
void reshape(GLFWwindow* window, int w, int h)
{
	mat4x4 projection, view;

	glViewport(0, 0, (GLsizei)w, (GLsizei)h);

	glMatrixMode(GL_PROJECTION);
	mat4x4_perspective(projection,
		2.f * (float)atan2(RADIUS, 200.0f),
		(float)w / (float)h,
		0.f, VIEW_SCENE_DIST);
	glLoadMatrixf((const GLfloat*)projection);

	glMatrixMode(GL_MODELVIEW);
	{
		vec3 eye = { 0.f, 0.f, VIEW_SCENE_DIST };
		vec3 center = { 0.f, 0.f, 0.f };
		vec3 up = { 0.f, -1.f, 0.f };
		mat4x4_look_at(view, eye, center, up);
	}
	glLoadMatrixf((const GLfloat*)view);
}

void key_callback(GLFWwindow* window, int key, int scancode, int action, int mods)
{
	std::cout << key << std::endl;
	if (key == 61){
		distance_change += 0.1;
	}
	else if (key == 45){
		distance_change -= 0.1;
	}
	else if (key == 119){
		translation.y -= 100;
	}
	else if (key == 115){
		translation.y += 100;
	}
	else if (key == 97){
		translation.x += 100;
	}
	else if (key == 100){
		translation.x -= 100;
	}

	if (key == GLFW_KEY_ESCAPE && action == GLFW_PRESS)
		glfwSetWindowShouldClose(window, GL_TRUE);
}

static void set_ball_pos(GLfloat x, GLfloat y)
{
	ball_x = (width / 2) - x;
	ball_y = y - (height / 2);
	dx = x_win_min - x;
	dy = y_win_min - y;

	float dx1 = -(float)(x - mouse_old_x)/100;
	float dy1 = (float)(y - mouse_old_y) / 100;



	rotate.x += -dy1 * 0.2f;
	rotate.y += dx1 * 0.2f;

	node_selected = closest_Node;

}

void mouse_button_callback(GLFWwindow* window, int button, int action, int mods)
{
	if (button != GLFW_MOUSE_BUTTON_LEFT)
		//override_pos = GL_TRUE;
		return;

	if (action == GLFW_PRESS)
	{
		override_pos = GL_TRUE;
		mouse_old_x = cursor_x;
		mouse_old_y = cursor_y;
		changeNode = true;
		set_ball_pos(cursor_x, cursor_y);
	}
	else
	{

		override_pos = GL_FALSE;
		changeNode = false;
		dx = dy = 0;


	}
}

void cursor_position_callback(GLFWwindow* window, double x, double y)
{
	cursor_x = (float)x;
	cursor_y = (float)y;
	mouse_old_x = cursor_x;
	mouse_old_x = cursor_y;
	//std::cout <<"coursor x " << cursor_x << std::endl;
	//std::cout << "coursor y " <<  cursor_y << std::endl;
	if (override_pos)
		set_ball_pos(cursor_x, cursor_y);
}

/*****************************************************************************
* Draw the Boing ball.
*
* The Boing ball is sphere in which each facet is a rectangle.
* Facet colors alternate between red and white.
* The ball is built by stacking latitudinal circles.  Each circle is composed
* of a widely-separated set of points, so that each facet is noticably large.
*****************************************************************************/
void DrawBoingBall(void)
{
	GLfloat lon_deg;     /* degree of longitude */
	double dt_total, dt2;

	glPushMatrix();
	glMatrixMode(GL_MODELVIEW);

	/*
	* Another relative Z translation to separate objects.
	*/
	glTranslatef(0.0, 0.0, DIST_BALL);

	/* Update ball position and rotation (iterate if necessary) */
	dt_total = dt;
	while (dt_total > 0.0)
	{
		dt2 = dt_total > MAX_DELTA_T ? MAX_DELTA_T : dt_total;
		dt_total -= dt2;
		BounceBall(dt2);
		deg_rot_y = TruncateDeg(deg_rot_y + deg_rot_y_inc*((float)dt2*ANIMATION_SPEED));
	}

	/* Set ball position */
	glTranslatef(ball_x, ball_y, 0.0);

	/*
	* Offset the shadow.
	*/
	if (drawBallHow == DRAW_BALL_SHADOW)
	{
		glTranslatef(SHADOW_OFFSET_X,
			SHADOW_OFFSET_Y,
			SHADOW_OFFSET_Z);
	}

	/*
	* Tilt the ball.
	*/
	glRotatef(-20.0, 0.0, 0.0, 1.0);

	/*
	* Continually rotate ball around Y axis.
	*/
	glRotatef(deg_rot_y, 0.0, 1.0, 0.0);

	/*
	* Set OpenGL state for Boing ball.
	*/
	glCullFace(GL_FRONT);
	glEnable(GL_CULL_FACE);
	glEnable(GL_NORMALIZE);

	/*
	* Build a faceted latitude slice of the Boing ball,
	* stepping same-sized vertical bands of the sphere.
	*/
	for (lon_deg = 0;
		lon_deg < 180;
		lon_deg += STEP_LONGITUDE)
	{
		/*
		* Draw a latitude circle at this longitude.
		*/
		DrawBoingBallBand(lon_deg,
			lon_deg + STEP_LONGITUDE);
	}

	glPopMatrix();

	return;
}


/*****************************************************************************
* Bounce the ball.
*****************************************************************************/
void BounceBall(double delta_t)
{
	GLfloat sign;
	GLfloat deg;

	if (override_pos)
		return;

	/* Bounce on walls */
	if (ball_x > (BOUNCE_WIDTH / 2 + WALL_R_OFFSET))
	{
		ball_x_inc = -0.5f - 0.75f * (GLfloat)rand() / (GLfloat)RAND_MAX;
		deg_rot_y_inc = -deg_rot_y_inc;
	}
	if (ball_x < -(BOUNCE_HEIGHT / 2 + WALL_L_OFFSET))
	{
		ball_x_inc = 0.5f + 0.75f * (GLfloat)rand() / (GLfloat)RAND_MAX;
		deg_rot_y_inc = -deg_rot_y_inc;
	}

	/* Bounce on floor / roof */
	if (ball_y > BOUNCE_HEIGHT / 2)
	{
		ball_y_inc = -0.75f - 1.f * (GLfloat)rand() / (GLfloat)RAND_MAX;
	}
	if (ball_y < -BOUNCE_HEIGHT / 2 * 0.85)
	{
		ball_y_inc = 0.75f + 1.f * (GLfloat)rand() / (GLfloat)RAND_MAX;
	}

	/* Update ball position */
	ball_x += ball_x_inc * ((float)delta_t*ANIMATION_SPEED);
	ball_y += ball_y_inc * ((float)delta_t*ANIMATION_SPEED);

	/*
	* Simulate the effects of gravity on Y movement.
	*/
	if (ball_y_inc < 0) sign = -1.0; else sign = 1.0;

	deg = (ball_y + BOUNCE_HEIGHT / 2) * 90 / BOUNCE_HEIGHT;
	if (deg > 80) deg = 80;
	if (deg < 10) deg = 10;

	ball_y_inc = sign * 4.f * (float)sin_deg(deg);
}


/*****************************************************************************
* Draw a faceted latitude band of the Boing ball.
*
* Parms:   long_lo, long_hi
*          Low and high longitudes of slice, resp.
*****************************************************************************/
void DrawBoingBallBand(GLfloat long_lo,
	GLfloat long_hi)
{
	vertex_t vert_ne;            /* "ne" means south-east, so on */
	vertex_t vert_nw;
	vertex_t vert_sw;
	vertex_t vert_se;
	vertex_t vert_norm;
	GLfloat  lat_deg;
	static int colorToggle = 0;

	/*
	* Iterate thru the points of a latitude circle.
	* A latitude circle is a 2D set of X,Z points.
	*/
	for (lat_deg = 0;
		lat_deg <= (360 - STEP_LATITUDE);
		lat_deg += STEP_LATITUDE)
	{
		/*
		* Color this polygon with red or white.
		*/
		if (colorToggle)
			glColor3f(0.8f, 0.1f, 0.1f);
		else
			glColor3f(0.95f, 0.95f, 0.95f);
#if 0
		if (lat_deg >= 180)
			if (colorToggle)
				glColor3f(0.1f, 0.8f, 0.1f);
			else
				glColor3f(0.5f, 0.5f, 0.95f);
#endif
		colorToggle = !colorToggle;

		/*
		* Change color if drawing shadow.
		*/
		if (drawBallHow == DRAW_BALL_SHADOW)
			glColor3f(0.35f, 0.35f, 0.35f);

		/*
		* Assign each Y.
		*/
		vert_ne.y = vert_nw.y = (float)cos_deg(long_hi) * RADIUS;
		vert_sw.y = vert_se.y = (float)cos_deg(long_lo) * RADIUS;

		/*
		* Assign each X,Z with sin,cos values scaled by latitude radius indexed by longitude.
		* Eg, long=0 and long=180 are at the poles, so zero scale is sin(longitude),
		* while long=90 (sin(90)=1) is at equator.
		*/
		vert_ne.x = (float)cos_deg(lat_deg) * (RADIUS * (float)sin_deg(long_lo + STEP_LONGITUDE));
		vert_se.x = (float)cos_deg(lat_deg) * (RADIUS * (float)sin_deg(long_lo));
		vert_nw.x = (float)cos_deg(lat_deg + STEP_LATITUDE) * (RADIUS * (float)sin_deg(long_lo + STEP_LONGITUDE));
		vert_sw.x = (float)cos_deg(lat_deg + STEP_LATITUDE) * (RADIUS * (float)sin_deg(long_lo));

		vert_ne.z = (float)sin_deg(lat_deg) * (RADIUS * (float)sin_deg(long_lo + STEP_LONGITUDE));
		vert_se.z = (float)sin_deg(lat_deg) * (RADIUS * (float)sin_deg(long_lo));
		vert_nw.z = (float)sin_deg(lat_deg + STEP_LATITUDE) * (RADIUS * (float)sin_deg(long_lo + STEP_LONGITUDE));
		vert_sw.z = (float)sin_deg(lat_deg + STEP_LATITUDE) * (RADIUS * (float)sin_deg(long_lo));

		/*
		* Draw the facet.
		*/
		glBegin(GL_POLYGON);

		CrossProduct(vert_ne, vert_nw, vert_sw, &vert_norm);
		glNormal3f(vert_norm.x, vert_norm.y, vert_norm.z);

		glVertex3f(vert_ne.x, vert_ne.y, vert_ne.z);
		glVertex3f(vert_nw.x, vert_nw.y, vert_nw.z);
		glVertex3f(vert_sw.x, vert_sw.y, vert_sw.z);
		glVertex3f(vert_se.x, vert_se.y, vert_se.z);

		glEnd();

#if BOING_DEBUG
		printf("----------------------------------------------------------- \n");
		printf("lat = %f  long_lo = %f  long_hi = %f \n", lat_deg, long_lo, long_hi);
		printf("vert_ne  x = %.8f  y = %.8f  z = %.8f \n", vert_ne.x, vert_ne.y, vert_ne.z);
		printf("vert_nw  x = %.8f  y = %.8f  z = %.8f \n", vert_nw.x, vert_nw.y, vert_nw.z);
		printf("vert_se  x = %.8f  y = %.8f  z = %.8f \n", vert_se.x, vert_se.y, vert_se.z);
		printf("vert_sw  x = %.8f  y = %.8f  z = %.8f \n", vert_sw.x, vert_sw.y, vert_sw.z);
#endif

	}

	/*
	* Toggle color so that next band will opposite red/white colors than this one.
	*/
	colorToggle = !colorToggle;

	/*
	* This circular band is done.
	*/
	return;
}


/*****************************************************************************
* Draw the purple grid of lines, behind the Boing ball.
* When the Workbench is dropped to the bottom, Boing shows 12 rows.
*****************************************************************************/
void DrawGrid(void)
{
	int              row, col;
	const int        rowTotal = 12;                   /* must be divisible by 2 */
	const int        colTotal = rowTotal;             /* must be same as rowTotal */
	const GLfloat    widthLine = 2.0;                  /* should be divisible by 2 */
	const GLfloat    sizeCell = GRID_SIZE / rowTotal;
	const GLfloat    z_offset = -40.0;
	GLfloat          xl, xr;
	GLfloat          yt, yb;

	glPushMatrix();
	glDisable(GL_CULL_FACE);

	/*
	* Another relative Z translation to separate objects.
	*/
	glTranslatef(0.0, 0.0, DIST_BALL);

	/*
	* Draw vertical lines (as skinny 3D rectangles).
	*/
	for (col = 0; col <= colTotal; col++)
	{
		/*
		* Compute co-ords of line.
		*/
		xl = -GRID_SIZE / 2 + col * sizeCell;
		xr = xl + widthLine;

		yt = GRID_SIZE / 2;
		yb = -GRID_SIZE / 2 - widthLine;

		glBegin(GL_POLYGON);

		glColor3f(0.2f, 0.1f, 0.6f);               /* purple */

		glVertex3f(xr, yt, z_offset);       /* NE */
		glVertex3f(xl, yt, z_offset);       /* NW */
		glVertex3f(xl, yb, z_offset);       /* SW */
		glVertex3f(xr, yb, z_offset);       /* SE */

		glEnd();
	}

	/*
	* Draw horizontal lines (as skinny 3D rectangles).
	*/
	for (row = 0; row <= rowTotal; row++)
	{
		/*
		* Compute co-ords of line.
		*/
		yt = GRID_SIZE / 2 - row * sizeCell;
		yb = yt - widthLine;

		xl = -GRID_SIZE / 2;
		xr = GRID_SIZE / 2 + widthLine;

		glBegin(GL_POLYGON);

		glColor3f(0.6f, 0.1f, 0.6f);               /* purple */

		glVertex3f(xr, yt, z_offset);       /* NE */
		glVertex3f(xl, yt, z_offset);       /* NW */
		glVertex3f(xl, yb, z_offset);       /* SW */
		glVertex3f(xr, yb, z_offset);       /* SE */

		glEnd();


		glBegin(GL_LINE_LOOP);

		glColor3f(0.6f, 0.1f, 0.6f);               /* purple */

		glVertex3f(35.0 - 200.0 + t, 359.0 - 200.0, z_offset);       /* NE */
		glVertex3f(353.0 - 200.0, 364.0 - 200.0, z_offset);       /* NW */
		glVertex3f(322.0 - 200.0, 34.0 - 200.0, z_offset);       /* SW */


		glEnd();
	}

	glPopMatrix();

	return;
}

void drawMesh(Geometry *p){
	int numElem = p->return_numElems();
	double x, y, z;
	double x_win, y_win;
	int node_considered = 0;
	double distance_old = INFINITY;
	double distance;
	int shortest_node = 0;
	for (int i = 0; i < numElem; i++){

		//glBegin(GL_TRIANGLES);

		//for (int j = 0; j < 4; j++){
		//	node_considered = p->node_number_inElem(i, j);
		//	x = p->return_x(node_considered);
		//	y = p->return_y(node_considered);
		//	z = p->return_z(node_considered);
		//	glColor3f(x, y, (float)i / numElem);
		//	glVertex3f(x * 400 - 200, y * 400 - 200, z*400);       /* NE */

		//	x_win = -(x * 400 - 400);
		//	y_win = y * 400;
		//	distance = ((x_win - cursor_x)*(x_win - cursor_x) + (y_win - cursor_y)*(y_win - cursor_y));
		//	if (distance_old > distance){
		//		distance_old = distance;
		//		shortest_node = node_considered;

		//	}
		int node_considered4 = 0;

		//}
		//glEnd();
		int node_considered1 = p->node_number_inElem(i, 0);
		int node_considered2 = p->node_number_inElem(i, 1);
		int node_considered3 = p->node_number_inElem(i, 2);
		if (p->return_dim() == 3){
			node_considered4 = p->node_number_inElem(i, 3);
		}
		
		
		
		
		x = p->return_x(node_considered1);
		y = p->return_y(node_considered1);
		if (p->return_dim()== 3){
			z = p->return_z(node_considered1);
		}
		
		if (p->return_dim() == 3){
			glLineWidth(2);
			glColor4f(1, y, z, 0.5);
			glBegin(GL_LINE_LOOP);
			glVertex3f(p->return_x(node_considered1) * 400 - 200, p->return_y(node_considered1) * 400 - 200, p->return_z(node_considered1) * 400);       /* NE */
			glVertex3f(p->return_x(node_considered2) * 400 - 200, p->return_y(node_considered2) * 400 - 200, p->return_z(node_considered2) * 400);       /* NE */
			glVertex3f(p->return_x(node_considered3) * 400 - 200, p->return_y(node_considered3) * 400 - 200, p->return_z(node_considered3) * 400);       /* NE */

			glEnd();

			glBegin(GL_LINE_LOOP);
			glVertex3f(p->return_x(node_considered2) * 400 - 200, p->return_y(node_considered2) * 400 - 200, p->return_z(node_considered2) * 400);
			glVertex3f(p->return_x(node_considered3) * 400 - 200, p->return_y(node_considered3) * 400 - 200, p->return_z(node_considered3) * 400);
			glVertex3f(p->return_x(node_considered4) * 400 - 200, p->return_y(node_considered4) * 400 - 200, p->return_z(node_considered4) * 400);
			glEnd();


			glBegin(GL_LINE_LOOP);
			glVertex3f(p->return_x(node_considered2) * 400 - 200, p->return_y(node_considered2) * 400 - 200, p->return_z(node_considered2) * 400);
			glVertex3f(p->return_x(node_considered4) * 400 - 200, p->return_y(node_considered4) * 400 - 200, p->return_z(node_considered4) * 400);
			glVertex3f(p->return_x(node_considered1) * 400 - 200, p->return_y(node_considered1) * 400 - 200, p->return_z(node_considered1) * 400);
			glEnd();

			glBegin(GL_LINE_LOOP);
			glVertex3f(p->return_x(node_considered1) * 400 - 200, p->return_y(node_considered1) * 400 - 200, p->return_z(node_considered1) * 400);
			glVertex3f(p->return_x(node_considered4) * 400 - 200, p->return_y(node_considered4) * 400 - 200, p->return_z(node_considered4) * 400);
			glVertex3f(p->return_x(node_considered3) * 400 - 200, p->return_y(node_considered3) * 400 - 200, p->return_z(node_considered3) * 400);
			glEnd();
		}
		else if (p->return_dim() == 2){
			//glColor4f(p->global_stress_mises[i]*10.0, 0.2, 0.5, 0.5);
			glColor3f(p->global_stress_mises[i] * 10.0, p->global_stress_mises[i] * 2.0, p->global_stress_mises[i] * 5.0);
			//glColor3f(1.0, 0.0, 1.0);
			glPolygonMode(GL_FRONT_AND_BACK, GL_TRIANGLES);
			glBegin(GL_TRIANGLES);
			glVertex3f(p->return_x(node_considered1) * 200 - 200, p->return_y(node_considered1) * 200 - 200, p->return_z(node_considered1) * 200);       /* NE */
			glVertex3f(p->return_x(node_considered2) * 200 - 200, p->return_y(node_considered2) * 200 - 200, p->return_z(node_considered2) * 200);       /* NE */
			glVertex3f(p->return_x(node_considered3) * 200 - 200, p->return_y(node_considered3) * 200 - 200, p->return_z(node_considered3) * 200);       /* NE */
			glColor3f(p->global_stress_mises[i] * 10.0, p->global_stress_mises[i] * 2.0, p->global_stress_mises[i] * 5.0);
			glEnd();
			glBegin(GL_LINE_LOOP);
			glVertex3f(p->return_x(node_considered1) * 200 - 200, p->return_y(node_considered1) * 200 - 200, p->return_z(node_considered1) * 200);
			glVertex3f(p->return_x(node_considered2) * 200 - 200, p->return_y(node_considered2) * 200 - 200, p->return_z(node_considered2) * 200);
			glVertex3f(p->return_x(node_considered3) * 200 - 200, p->return_y(node_considered3) * 200 - 200, p->return_z(node_considered3) * 200);
			glEnd();
			
		}
		x_win = -(x * 400 - 400);
			y_win = y * 400;
			distance = ((x_win - cursor_x)*(x_win - cursor_x) + (y_win - cursor_y)*(y_win - cursor_y));
			if (distance_old > distance){
				distance_old = distance;
				shortest_node = node_considered;

		}
	
		//glColor4f(x, y, (float)i / numElem,0.5 );
		//glColor3f(1.0, 0.0, 1.0);
		//glPolygonMode(GL_FRONT_AND_BACK, GL_TRIANGLES);
		//glBegin(GL_TRIANGLES);
		//glVertex3f(p->return_x(node_considered1) * 400 - 200, p->return_y(node_considered1) * 400 - 200, p->return_z(node_considered1) * 400-200);       /* NE */
		//glVertex3f(p->return_x(node_considered2) * 400 - 200, p->return_y(node_considered2) * 400 - 200, p->return_z(node_considered2) * 400 - 200);       /* NE */
		//glVertex3f(p->return_x(node_considered3) * 400 - 200, p->return_y(node_considered3) * 400 - 200, p->return_z(node_considered3) * 400 - 200);       /* NE */

		//glEnd();

		//glBegin(GL_TRIANGLES);
		//glVertex3f(p->return_x(node_considered2) * 400 - 200, p->return_y(node_considered2) * 400 - 200, p->return_z(node_considered2) * 400 - 200);
		//glVertex3f(p->return_x(node_considered3) * 400 - 200, p->return_y(node_considered3) * 400 - 200, p->return_z(node_considered3) * 400 - 200);
		//glVertex3f(p->return_x(node_considered4) * 400 - 200, p->return_y(node_considered4) * 400 - 200, p->return_z(node_considered4) * 400 - 200);
		//glEnd();

		//
		//glBegin(GL_TRIANGLES);
		//glVertex3f(p->return_x(node_considered2) * 400 - 200, p->return_y(node_considered2) * 400 - 200, p->return_z(node_considered2) * 400 - 200);
		//glVertex3f(p->return_x(node_considered4) * 400 - 200, p->return_y(node_considered4) * 400 - 200, p->return_z(node_considered4) * 400 - 200);
		//glVertex3f(p->return_x(node_considered1) * 400 - 200, p->return_y(node_considered1) * 400 - 200, p->return_z(node_considered1) * 400 - 200);
		//glEnd();

		//glBegin(GL_TRIANGLES);
		//glVertex3f(p->return_x(node_considered1) * 400 - 200, p->return_y(node_considered1) * 400 - 200, p->return_z(node_considered1) * 400 - 200);
		//glVertex3f(p->return_x(node_considered4) * 400 - 200, p->return_y(node_considered4) * 400 - 200, p->return_z(node_considered4) * 400 - 200);
		//glVertex3f(p->return_x(node_considered3) * 400 - 200, p->return_y(node_considered3) * 400 - 200, p->return_z(node_considered3) * 400 - 200);
		//glEnd();
		



		//glBegin(GL_LINE_LOOP);
		//glColor3f(1.0, 1.0, 0.0);
		//for (int j = 0; j < 4; j++){
		//	node_considered = p->node_number_inElem(i, j);
		//	x = p->return_x(node_considered);
		//	y = p->return_y(node_considered);
		//	z = p->return_z(node_considered);
		//	glVertex3f(x * 400 - 200, y * 400 - 200, z*400);       /* NE */

		//	x_win = -(x * 400 - 400);
		//	y_win = y * 400;
		//	distance = ((x_win - cursor_x)*(x_win - cursor_x) + (y_win - cursor_y)*(y_win - cursor_y));
		//	if (distance_old > distance){
		//		distance_old = distance;
		//		shortest_node = node_considered;

		//	}


		//}
		//glEnd();
	}
	glColor3f(0.6f, 1.0f, 0.6f);
	glPointSize(10.0);

	
	glBegin(GL_POINTS);
	glVertex3f(p->return_x(0) * 400 - 200, p->return_y(0) * 400 - 200, p->return_z(0)*400 );

	glColor3f(0.0f, 1.0f, 1.6f);
	glVertex3f(p->return_x(20) * 200 - 200, p->return_y(20) * 200 - 200, p->return_z(20) * 400);

	glEnd();

	//std::cout << " X _ win: " << -(p->return_x(shortest_node) * 400-400) << " Y _ WIN : " << p->return_y(shortest_node) * 400 << std::endl;
	if (!changeNode){
		closest_Node = shortest_node;
	}
	x_win_min = -(p->return_x(closest_Node) * 400 - 400);
	y_win_min = p->return_y(closest_Node) * 400;

}

/*======================================================================*
* main()
*======================================================================*/

int draw_things(Geometry *p)
{
	GLFWwindow* window;

	/* Init GLFW */
	if (!glfwInit())
		exit(EXIT_FAILURE);

	glfwWindowHint(GLFW_DEPTH_BITS, 16);

	window = glfwCreateWindow(400, 400, "FEM TEST", NULL, NULL);
	if (!window)
	{
		glfwTerminate();
		exit(EXIT_FAILURE);
	}

	glfwSetFramebufferSizeCallback(window, reshape);
	glfwSetKeyCallback(window, key_callback);
	glfwSetMouseButtonCallback(window, mouse_button_callback);
	glfwSetCursorPosCallback(window, cursor_position_callback);

	glfwMakeContextCurrent(window);
	glfwSwapInterval(1);

	glfwGetFramebufferSize(window, &width, &height);
	reshape(window, width, height);

	glfwSetTime(0.0);

	init();
	t = 0;
	/* Main loop */
	//p->initilizeMatrices();

	//----------opencmiss



	glfwMakeContextCurrent(window);

	glfwPollEvents();

	///-cmiss
	double duration_K;
	bool cuda_init = false;
	int display_counter = 0;
	//initilizing all of the vectors
	if (p->get_dynamic())
		p->initialize_dynamic();
	p->set_beta1(0.9); // if beta_2 >= beta1 and beta > 1/2 then the time stepping scheme is unconditionally stable.
	p->set_beta2(0.9);
	p->set_dt(0.05);
	p->set_dynamic_alpha(0.2);
	p->set_dynamic_xi(0.23);
	p->initialize_zerovector(9);
	//next we set what nodes we want to make stable
	int points[9];
	for (int i = 0; i < 9; i++){
		points[i] = i;

	}

	p->set_zero_nodes(points);
	if (!p->get_dynamic()){
		for (;;){



			///* Timing */
			//t = glfwGetTime();
			//dt = t - t_old;
			//t_old = t;
			//glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
			//glPushMatrix();
			//glRotatef(rotate.x, 1.0, 0.0, 0.0);
			//glRotatef(rotate.y, 0.0, 100.0, 0.0);
			//glScalef(distance_change, distance_change, distance_change);
			//glTranslatef(translation.x, translation.y, translation.z);
			///* Draw one frame */
			////display();
			////DrawGrid();
			//drawMesh(p);
			//glPopMatrix();
			//glFlush();
			///* Swap buffers */
			//glfwSwapBuffers(window);
			//glfwPollEvents();



			////Solve the 2D FEM in each frame
			//if (display_counter < 100){
			//	p->setSudoNode(20);
			//	p->setSudoForcex(0);
			//	p->setSudoForcey(0);
			//}
			//else {
			//	p->setSudoNode(20);
			//	p->setSudoForcex(0);
			//	p->setSudoForcey(0);
			//}


			//display_counter++;
			//if (p->return_dim() == 3){
			//	p->Linear3DBarycentric_B_CUDA_host();
			//}


			//p->make_K_matrix();

			////p->make_surface_f();



			//if (!cuda_init){
			//	p->initialize_CUDA();

			//	cuda_init = true;
			//}
			//std::clock_t start_K;
			//start_K = std::clock();
			//p->tt();
			//duration_K = (std::clock() - start_K) / (double)CLOCKS_PER_SEC;


			////std::cout << " change status : " << changeNode << std::endl;

			//std::cout << "Solver time ms:  " << duration_K << std::endl;
			////std::cout << " closet node : " << closest_Node << std::endl;
			//t++;


			/* Timing */
			t = glfwGetTime();
			dt = t - t_old;
			t_old = t;
			glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
			glPushMatrix();
			glRotatef(rotate.x, 1.0, 0.0, 0.0);
			glRotatef(rotate.y, 0.0, 100.0, 0.0);
			glScalef(distance_change, distance_change, distance_change);
			glTranslatef(translation.x, translation.y, translation.z);
			/* Draw one frame */
			//display();
			//DrawGrid();
			drawMesh(p);
			


			////////-----------------------CMISS

		

			////////////-------------------------------
			glPopMatrix();
			glFlush();
			
			/* Swap buffers */
			glfwSwapBuffers(window);
			
			glfwPollEvents();
			
			std::clock_t start_K;
			start_K = std::clock();

			//Solve the 2D FEM in each frame
			p->setSudoNode(200);
			p->setSudoForcex( 6.0);
			p->setSudoForcey( 6.0);
			
			/*if (p->return_dim() == 3){
				p->Linear3DBarycentric_B_CUDA_host();
			}
*/

			p->make_K_matrix();

			//p->make_surface_f();



			if (!cuda_init){
				p->initialize_CUDA();
				cuda_init = true;
			}
		
			p->tt();
			duration_K = (std::clock() - start_K) / (double)CLOCKS_PER_SEC;


			//std::cout << " change status : " << changeNode << std::endl;

			std::cout << "Solver time ms:  " << duration_K << std::endl;
			//std::cout << " closet node : " << closest_Node << std::endl;
			t++;
			/* Check if we are still running */
			if (glfwWindowShouldClose(window))
				break;
		}
	}
	else{
		for (;;){
			if (display_counter < 1){
				p->setSudoNode(900);
				p->setSudoForcex(3000.0);
				p->setSudoForcey(2000.0);
			}
			else {
				p->setSudoNode(120);
				p->setSudoForcex(0);
				p->setSudoForcey(0);
			}
		
			/*if (display_counter == 500){
				p->setSudoNode(20);
				p->setSudoForcex(-100);
				p->setSudoForcey(-100);
			}*/
			display_counter++;
			t = glfwGetTime();
			dt = t - t_old;
			t_old = t;
			glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
			glPushMatrix();
			glRotatef(rotate.x, 1.0, 0.0, 0.0);
			glRotatef(rotate.y, 0.0, 100.0, 0.0);
			glScalef(distance_change, distance_change, distance_change);
			glTranslatef(translation.x, translation.y, translation.z);
			/* Draw one frame */
			//display();
			//DrawGrid();
			drawMesh(p);
			glPopMatrix();
			glFlush();
			/* Swap buffers */
			glfwSwapBuffers(window);
			glfwPollEvents();

			///dynamic calculations
		
			p->make_K_matrix();
			
			p->find_b();
			std::clock_t start_K;
			start_K = std::clock();
			p->update_vector();
			duration_K = (std::clock() - start_K) / (double)CLOCKS_PER_SEC;
			std::cout << "Solver time ms:  " << duration_K << std::endl;
			p->update_dynamic_vectors();
			p->update_dynamic_xyz();
			t++;
			
			/* Check if we are still running */
			if (glfwWindowShouldClose(window))
				break;
		}
	}

	glfwTerminate();
	exit(EXIT_SUCCESS);
}
